/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <layers/matrix_multiply_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {
template <typename T>
MatrixMultiplyLayer<T>::MatrixMultiplyLayer(const std::vector<core23::Tensor>& input_tensors,
                                            core23::Tensor& output_tensor,
                                            const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer({}, {}, gpu_resource) {
  try {
    num_ = input_tensors.size();

    // error input checking
    dims_ = input_tensors[0].shape().dims();
    if (num_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer needs at least 2 input tensors");
    }
    if (input_tensors[1].shape().dims() != (int64_t)dims_) {
      HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same num of dims");
    }
    if (input_tensors[1].shape().size(dims_ - 2) != input_tensors[0].shape().size(dims_ - 1)) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "The last two dimension of the input tensors should be m x n, n x k");
    }

    for (size_t i = 0; i < num_; i++) {
      input_tensors_.push_back(input_tensors[i]);
    }

    int64_t m = input_tensors[0].shape().size(dims_ - 2);
    int64_t k = input_tensors[1].shape().size(dims_ - 1);

    core23::TensorParams out_params = input_tensors[0].my_params();
    if (dims_ == 2) {
      std::vector<int64_t> out_shape = {m, k};
      output_tensor = core23::Tensor(out_params.shape(out_shape));
    } else if (dims_ == 3) {  // dims_ == 3
      if (input_tensors[0].shape().size(0) != input_tensors[1].shape().size(0)) {
        HCTR_OWN_THROW(Error_t::WrongInput, "3D input tensors must have the same batch size");
      }
      int64_t b = input_tensors[0].shape().size(0);
      std::vector<int64_t> out_shape = {b, m, k};
      output_tensor = core23::Tensor(out_params.shape(out_shape));
    } else if (dims_ == 4) {
      if (input_tensors[0].shape().size(0) != input_tensors[1].shape().size(0)) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same batch size");
      }
      if (input_tensors[0].shape().size(1) != input_tensors[1].shape().size(1)) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same second dim");
      }
      int64_t b = input_tensors[0].shape().size(0);
      int64_t num_head = input_tensors[0].shape().size(1);
      std::vector<int64_t> out_shape = {b, num_head, m, k};
      output_tensor = core23::Tensor(out_params.shape(out_shape));
    }

    output_tensors_.push_back(output_tensor);

    fprop_inputA_tensor23_ = core23::Tensor(out_params);

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
MatrixMultiplyLayer<T>::MatrixMultiplyLayer(
    const Tensors2<T>& in_tensors, Tensor2<T>& out_tensor,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  try {
    num_ = in_tensors.size();

    // error input checking
    dims_ = in_tensors[0].get_dimensions().size();
    if (num_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer needs at least 2 input tensors");
    }
    if (in_tensors[1].get_dimensions().size() != dims_) {
      HCTR_OWN_THROW(Error_t::WrongInput, "All the input tensors must have the same num of dims");
    }
    if (in_tensors[1].get_dimensions()[dims_ - 2] != in_tensors[0].get_dimensions()[dims_ - 1]) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "The last two dimension of the input tensors should be m x n, n x k");
    }

    for (size_t i = 0; i < num_; i++) {
      in_tensors_.push_back(in_tensors[i]);
    }

    size_t m = in_tensors[0].get_dimensions()[dims_ - 2];
    size_t k = in_tensors[1].get_dimensions()[dims_ - 1];

    if (dims_ == 2) {
      std::vector<size_t> out_dim = {m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    } else if (dims_ == 3) {  // dims_ == 3
      if (in_tensors[0].get_dimensions()[0] != in_tensors[1].get_dimensions()[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "3D input tensors must have the same batch size");
      }
      size_t b = in_tensors[0].get_dimensions()[0];
      std::vector<size_t> out_dim = {b, m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    } else if (dims_ == 4) {
      if (in_tensors[0].get_dimensions()[0] != in_tensors[1].get_dimensions()[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same batch size");
      }
      if (in_tensors[0].get_dimensions()[1] != in_tensors[1].get_dimensions()[1]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same second dim");
      }
      size_t b = in_tensors[0].get_dimensions()[0];
      size_t num_head = in_tensors[0].get_dimensions()[1];
      std::vector<size_t> out_dim = {b, num_head, m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    }

    out_tensors_.push_back(out_tensor);

    blobs_buff->reserve(in_tensors[0].get_dimensions(), &fprop_inputA_);

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void MatrixMultiplyLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  if (input_tensors_.empty()) {
    T* in1 = in_tensors_[0].get_ptr();
    T* in2 = in_tensors_[1].get_ptr();
    T* out = out_tensors_[0].get_ptr();

    const auto& in_tensor_dim = in_tensors_[0].get_dimensions();
    const auto& out_tensor_dim = out_tensors_[0].get_dimensions();

    size_t m, n, k, b = 1;

    b = dims_ == 3 ? in_tensor_dim[0] : 1;
    b = dims_ == 4 ? in_tensor_dim[0] * in_tensor_dim[1] : b;
    m = in_tensor_dim[dims_ - 2];
    n = in_tensor_dim[dims_ - 1];
    k = out_tensor_dim[dims_ - 1];
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (size_t i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_out, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }

    HCTR_LIB_THROW(hipMemcpyAsync((void*)fprop_inputA_.get_ptr(), (void*)in1,
                                   in_tensors_[0].get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                   get_gpu().get_stream()));
  } else {
    T* in1 = input_tensors_[0].data<T>();
    T* in2 = input_tensors_[1].data<T>();
    T* out = output_tensors_[0].data<T>();

    const auto& input_tensor_shape = input_tensors_[0].shape();
    const auto& output_tensor_shape = output_tensors_[0].shape();

    int64_t m, n, k, b = 1;

    b = dims_ == 3 ? input_tensor_shape.size(0) : 1;
    b = dims_ == 4 ? input_tensor_shape.size(0) * input_tensor_shape.size(1) : b;
    m = input_tensor_shape.size(dims_ - 2);
    n = input_tensor_shape.size(dims_ - 1);
    k = output_tensor_shape.size(dims_ - 1);
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (auto i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_out, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }

    HCTR_LIB_THROW(hipMemcpyAsync(fprop_inputA_tensor23_.data(), (void*)in1,
                                   input_tensors_[0].num_bytes(), hipMemcpyDeviceToDevice,
                                   get_gpu().get_stream()));
  }
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template <typename T>
void MatrixMultiplyLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  if (input_tensors_.empty()) {
    T* in1 = in_tensors_[0].get_ptr();
    T* in2 = in_tensors_[1].get_ptr();
    T* out = out_tensors_[0].get_ptr();

    const auto& in_tensor_dim = in_tensors_[0].get_dimensions();
    const auto& out_tensor_dim = out_tensors_[0].get_dimensions();

    size_t m, n, k, b = 1;

    b = dims_ == 3 ? in_tensor_dim[0] : 1;
    b = dims_ == 4 ? in_tensor_dim[0] * in_tensor_dim[1] : b;
    m = in_tensor_dim[dims_ - 2];
    n = in_tensor_dim[dims_ - 1];
    k = out_tensor_dim[dims_ - 1];
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (size_t i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      // gradient respect to A
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_out, HIP_R_32F, k, &beta,
                                  cur_in1, HIP_R_32F, n, compute_type, HIPBLAS_GEMM_DEFAULT));

      cur_in1 = fprop_inputA_.get_ptr() + i * m * n;
      // gradient respect to B
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m,
                                  &alpha, cur_out, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_in2, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }
  } else {
    T* in1 = input_tensors_[0].data<T>();
    T* in2 = input_tensors_[1].data<T>();
    T* out = output_tensors_[0].data<T>();

    const auto& input_tensor_shape = input_tensors_[0].shape();
    const auto& output_tensor_shape = output_tensors_[0].shape();

    int64_t m, n, k, b = 1;

    b = dims_ == 3 ? input_tensor_shape.size(0) : 1;
    b = dims_ == 4 ? output_tensor_shape.size(0) * input_tensor_shape.size(1) : b;
    m = input_tensor_shape.size(dims_ - 2);
    n = input_tensor_shape.size(dims_ - 1);
    k = output_tensor_shape.size(dims_ - 1);
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (auto i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      // gradient respect to A
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_out, HIP_R_32F, k, &beta,
                                  cur_in1, HIP_R_32F, n, compute_type, HIPBLAS_GEMM_DEFAULT));

      cur_in1 = fprop_inputA_tensor23_.data<T>() + i * m * n;
      // gradient respect to B
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m,
                                  &alpha, cur_out, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_in2, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }
  }
#ifndef NDEBUG
  hipDeviceSynchronize();
  HCTR_LIB_THROW(hipGetLastError());
#endif
}

template class MatrixMultiplyLayer<float>;

}  // namespace HugeCTR
