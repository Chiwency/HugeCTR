#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <collectives/all_reduce_comm.hpp>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/calibration_data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/indices_container.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/statistics.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/embeddings/hybrid_sparse_embedding.hpp"
#include "HugeCTR/include/tensor2.hpp"

namespace HugeCTR {
template <typename dtype, typename emtype>
HybridSparseEmbedding<dtype, emtype>::HybridSparseEmbedding(
    const SparseTensors<dtype> &train_input_tensors,
    const SparseTensors<dtype> &evaluate_input_tensors,
    const HybridSparseEmbeddingParams &embedding_params,
    const std::vector<BuffPtr<emtype>> &grouped_wgrad_buff,
    const GpuLearningRateSchedulers lr_scheds, bool graph_mode,
    const std::shared_ptr<ResourceManager> &resource_manager, bool overlap_ar_a2a,
    bool eval_overlap)
    : train_input_tensors_(train_input_tensors),
      evaluate_input_tensors_(evaluate_input_tensors),
      embedding_params_(embedding_params),
      resource_manager_(resource_manager),
      stream_manager_(resource_manager->get_local_gpu_count()),
      grouped_wgrad_buff_(grouped_wgrad_buff),
      grouped_all_reduce_(grouped_wgrad_buff[0] != NULL),
      lr_scheds_(lr_scheds),
      graph_mode_(graph_mode),
      overlap_ar_a2a_(overlap_ar_a2a),
      eval_overlap_(eval_overlap) {
  try {
    // 0. Error check
    if (embedding_params_.train_batch_size < 1 || embedding_params_.evaluate_batch_size < 1 ||
        embedding_params_.slot_num < 1 || embedding_params_.embedding_vec_size < 1) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "batchsize < 1 || slot_num < 1 || embedding_vec_size < 1");
    }

    if (embedding_params_.embedding_vec_size > 1024) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "the embedding_vec_size can not be more than 1024 in embedding layer");
    }

    size_t total_gpu_count = resource_manager_->get_global_gpu_count();
    size_t local_gpu_count = resource_manager_->get_local_gpu_count();

    if (train_input_tensors.size() != local_gpu_count ||
        evaluate_input_tensors.size() != local_gpu_count) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "either train_input_tensors.size() or evaluate_input_tensors.size() isn't "
                     "local_gpu_count_");
    }

    HCTR_LOG_S(INFO, ROOT) << "Using Hybrid Embedding with train batch " << get_batch_size(true)
                           << " and eval batch " << get_batch_size(false) << std::endl;

    // 1. initialize optimizer
    for (size_t id = 0; id < local_gpu_count; id++) {
      OptParams opt_params;
      opt_params.optimizer = embedding_params_.opt_params.optimizer;
      opt_params.lr = embedding_params_.opt_params.lr;
      opt_params.update_type = embedding_params_.opt_params.update_type;
      opt_params.scaler = embedding_params_.opt_params.scaler;
      opt_params_.emplace_back(opt_params);
    }
    // 2. reserve buffers for different tensors
    data_statistics_.reserve(local_gpu_count);
    data_train_.reserve(local_gpu_count);
    data_evaluate_.reserve(local_gpu_count);
    model_.reserve(local_gpu_count);
    calibration_.reserve(local_gpu_count);
    statistics_.reserve(local_gpu_count);
    train_output_tensors_.reserve(local_gpu_count);
    evaluate_output_tensors_.reserve(local_gpu_count);
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      frequent_embeddings_single_node_.reserve(local_gpu_count);
    } else {
      frequent_embeddings_multi_node_.reserve(local_gpu_count);
    }

    infrequent_embeddings_single_node_.reserve(local_gpu_count);
    infrequent_embeddings_ib_nvlink_.reserve(local_gpu_count);
    infrequent_embeddings_ib_nvlink_hier_.reserve(local_gpu_count);

    assert(bufs_.empty());
    CudaDeviceContext context;
    // 2.1. construct data
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      data_statistics_.emplace_back(embedding_params_.slot_size_array, get_batch_size(true),
                                    embedding_params_.num_iterations_statistics);
      if (!embedding_params_.use_train_precompute_indices) {
        data_train_.emplace_back(embedding_params_.slot_size_array, get_batch_size(true), 1);
      }
      if (!embedding_params_.use_eval_precompute_indices) {
        data_evaluate_.emplace_back(embedding_params_.slot_size_array, get_batch_size(false), 1);
      }
    }

    // 2.2 construct model
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      std::vector<uint32_t> num_instances_per_node(resource_manager_->get_num_process(), 0);
      get_num_instances_per_node(num_instances_per_node);
      model_.emplace_back(embedding_params_.communication_type,
                          resource_manager_->get_local_gpu(i)->get_global_id(),
                          num_instances_per_node, get_categories_num());
    }

    // 2.3 construct calibration
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      calibration_.emplace_back(resource_manager_->get_num_process(), embedding_params_.p_dup_max,
                                embedding_params_.max_all_reduce_bandwidth,
                                embedding_params_.max_all_to_all_bandwidth,
                                embedding_params_.efficiency_bandwidth_ratio);
    }

    // 2.4 construct Statistics
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      const size_t num_samples_statistics = embedding_params_.num_iterations_statistics *
                                            get_batch_size(true) * embedding_params_.slot_num;
      statistics_.emplace_back((dtype)num_samples_statistics, embedding_params_.slot_num,
                               model_[i].num_instances, get_categories_num());
    }

    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      std::shared_ptr<GeneralBuffer2<CudaAllocator>> buf = GeneralBuffer2<CudaAllocator>::create();
      bufs_.emplace_back(buf);
      // 2.5. reserve for train output/ evaluate output tensors
      Tensor2<emtype> tensor;
      buf->reserve({get_batch_size_per_gpu(true), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      train_output_tensors_.emplace_back(tensor);
      buf->reserve({get_batch_size_per_gpu(false), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      evaluate_output_tensors_.emplace_back(tensor);

      // 2.6 construct frequent embedding
      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        frequent_embeddings_single_node_.emplace_back(
            model_[i], get_local_gpu(i), grouped_wgrad_buff_[i], get_embedding_vec_size(),
            embedding_params_.max_num_frequent_categories);
      } else {
        frequent_embeddings_multi_node_.emplace_back(
            model_[i], get_local_gpu(i), grouped_wgrad_buff_[i], get_embedding_vec_size(),
            embedding_params_.max_num_frequent_categories);
      }

      if (!embedding_params_.use_train_precompute_indices) {
        frequent_embedding_train_indices_.emplace_back(
            embedding_params_.max_num_frequent_categories, data_train_[i], model_[i]);
      }
      if (!embedding_params_.use_eval_precompute_indices) {
        frequent_embedding_evaluate_indices_.emplace_back(
            embedding_params_.max_num_frequent_categories, data_evaluate_[i], model_[i]);
      }

      // 2.7 construct infrequent embedding
      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        infrequent_embeddings_single_node_.emplace_back(model_[i], get_local_gpu(i),
                                                        get_embedding_vec_size());
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        infrequent_embeddings_ib_nvlink_.emplace_back(model_[i], get_local_gpu(i),
                                                      get_embedding_vec_size());
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_.emplace_back(model_[i], get_local_gpu(i),
                                                           get_embedding_vec_size());
      }

      if (!embedding_params_.use_train_precompute_indices) {
        infrequent_embedding_train_indices_.emplace_back(data_train_[i], model_[i]);
      }
      if (!embedding_params_.use_eval_precompute_indices) {
        infrequent_embedding_evaluate_indices_.emplace_back(data_evaluate_[i], model_[i]);
      }

      // 2.8 construct communication
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        size_t max_buf_size = embedding_params_.embedding_vec_size *
                              std::max(get_batch_size(true), get_batch_size(false)) *
                              embedding_params_.slot_num;
        infrequent_embeddings_ib_nvlink_.back().init_comms(
            embedding_params_.embedding_vec_size, &get_local_gpu(i), buf.get(), max_buf_size);
      }

      // Construct comm buffers
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_[i].init_comms(
            embedding_params_.max_num_infrequent_samples, embedding_params_.slot_num,
            embedding_params_.embedding_vec_size, buf.get(), get_batch_size(true),
            get_batch_size(false), local_gpu_count);
      }

      // For global barrier in eval
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1}, &tensor);
        d_barrier_store_.push_back(tensor);
      }
      buf->allocate();
    }

    // Frequent AR comm init
    if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
        (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
      if (!grouped_all_reduce_) {
        // Do your own all-reduce
        auto ar_comm = resource_manager_->get_ar_comm();
        frequent_embedding_handle_ = ar_comm->register_coll();
        // Frequent all reduce comm
        for (uint32_t i = 0; i < local_gpu_count; i++) {
          frequent_embeddings_multi_node_[i].init_ar_comm(ar_comm, frequent_embedding_handle_, i);
        }
        ar_comm->register_coll_buf(frequent_embedding_handle_);
      }
    }

    // Init after buffer allocation
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI
      ib_comm_ = resource_manager_->get_ib_comm();
      if (!ib_comm_) {
        resource_manager_->init_ib_comm();
        ib_comm_ = resource_manager_->get_ib_comm();
      }
      comm_stream_.resize(local_gpu_count);

      std::vector<size_t *> h_model_indices_sizes_ptrs(local_gpu_count);
      std::vector<size_t *> h_network_indices_sizes_ptrs(local_gpu_count);
      std::vector<emtype *> h_fwd_send_buffer_ptrs(local_gpu_count);
      std::vector<emtype *> h_bwd_send_buffer_ptrs(local_gpu_count);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        h_model_indices_sizes_ptrs[i] =
            infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_.get_ptr();
        h_network_indices_sizes_ptrs[i] =
            infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_.get_ptr();
        h_fwd_send_buffer_ptrs[i] = infrequent_embeddings_ib_nvlink_hier_[i]
                                        .infrequent_forward_comm_buffers_->send_buffer.get_ptr();
        h_bwd_send_buffer_ptrs[i] = infrequent_embeddings_ib_nvlink_hier_[i]
                                        .infrequent_backward_comm_buffers_->send_buffer.get_ptr();
      }

      // Forward coll init
      auto infrequent_forward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        // download pointers
        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_ptrs_.get_ptr(),
            h_model_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_ptrs_.get_ptr(),
            h_network_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_ib_nvlink_hier_[i]
                                .infrequent_forward_comm_buffers_->send_buffer_ptrs.get_ptr(),
                            h_fwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_ib_nvlink_hier_[i]
                                .infrequent_backward_comm_buffers_->send_buffer_ptrs.get_ptr(),
                            h_bwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipStreamSynchronize(get_local_gpu(i).get_stream()));

        // Initialize IB comm
        HCTR_LIB_THROW(hipStreamCreateWithPriority(&comm_stream_[i], hipStreamNonBlocking, -100));
        ib_comm_->set_a2a_coll_stream(infrequent_forward_coll_handle, comm_stream_[i], i);

        ib_comm_->set_a2a_coll_buf(
            infrequent_forward_coll_handle,
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->send_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->send_buffer.get_size_in_bytes(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->recv_buffer.get_size_in_bytes(),
            i);

        infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_ =
            std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
                i, infrequent_forward_coll_handle,
                infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_ptrs_.get_ptr(),
                &get_local_gpu(i), ib_comm_, comm_stream_[i]);
      }
      ib_comm_->register_a2a_coll_buf(infrequent_forward_coll_handle);

      // Backward coll init
      auto infrequent_backward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        ib_comm_->set_a2a_coll_stream(infrequent_backward_coll_handle, comm_stream_[i], i);
        ib_comm_->set_a2a_coll_buf(
            infrequent_backward_coll_handle,
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->send_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->send_buffer.get_size_in_bytes(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->recv_buffer.get_size_in_bytes(),
            i);

        infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_ =
            std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
                i, infrequent_backward_coll_handle,
                infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_ptrs_.get_ptr(),
                &get_local_gpu(i), ib_comm_, comm_stream_[i]);
      }
      ib_comm_->register_a2a_coll_buf(infrequent_backward_coll_handle);
#else
      HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
    }

    // 2.9 Single-node: copy some pointers arrays to device
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      // Initialize GPU barrier
      gpu_barrier_ = std::make_unique<GPUBarrier>(resource_manager_->get_local_gpu_count(),
                                                  resource_manager_->get_local_gpu_device_id_list(),
                                                  graph_mode_);

      std::vector<const emtype *> frequent_vectors_cache_pointers(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_train(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_eval(local_gpu_count);
      std::vector<const emtype *> gradients_pointers(local_gpu_count);
      std::vector<const emtype *> frequent_partial_gradients_pointers(local_gpu_count);

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        frequent_vectors_cache_pointers[i] =
            frequent_embeddings_single_node_[i].get_embedding_vectors_cache().get_ptr();
        interaction_layer_input_pointers_train[i] = train_output_tensors_[i].get_ptr();
        gradients_pointers[i] = train_output_tensors_[i].get_ptr();
        interaction_layer_input_pointers_eval[i] = evaluate_output_tensors_[i].get_ptr();
        frequent_partial_gradients_pointers[i] =
            frequent_embeddings_single_node_[i].frequent_data_.get_gradients().get_ptr();
      }

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_single_node_[i].embedding_vectors_cache_pointers_.get_ptr(),
            frequent_vectors_cache_pointers.data(), local_gpu_count * sizeof(float *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        infrequent_embeddings_single_node_[i].init_pointers(
            local_gpu_count, get_local_gpu(i).get_stream(), interaction_layer_input_pointers_train,
            interaction_layer_input_pointers_eval, gradients_pointers);
        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_single_node_[i].partial_gradients_pointers_.get_ptr(),
            frequent_partial_gradients_pointers.data(), local_gpu_count * sizeof(emtype *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
      }
    }
  } catch (const std::runtime_error &rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_model(const SparseTensors<dtype> &data,
                                                      size_t &wgrad_offset_in_bytes) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();
  HCTR_LOG(INFO, ROOT, "Initializing Hybrid Embedding\n");
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t id = 0; id < local_gpu_count; ++id) {
    int cur_device = get_local_gpu(id).get_device_id();
    CudaDeviceContext context(cur_device);
    auto stream = get_local_gpu(id).get_stream();
    data_statistics_[id].data_to_unique_categories(data[id].get_value_tensor(), stream);
    model_[id].init_hybrid_model(calibration_[id], statistics_[id], data_statistics_[id], stream);

    get_frequent_embedding_data(id).initialize_embedding_vectors(data_statistics_[id].table_sizes,
                                                                 wgrad_offset_in_bytes);

    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      infrequent_embeddings_single_node_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
      infrequent_embeddings_ib_nvlink_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
      infrequent_embeddings_ib_nvlink_hier_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }

    if (embedding_params_.max_num_frequent_categories < (size_t)model_[id].num_frequent) {
      HCTR_OWN_THROW(
          Error_t::WrongInput,
          "Found too many frequent categories, please increase 'max_num_frequent_categories'");
    }
  }

  HCTR_LOG_S(INFO, ROOT) << "Initialized hybrid model with " << model_[0].num_frequent
                         << " frequent categories, probability of being frequent is "
                         << model_[0].frequent_probability << std::endl;

  size_t avg_train_infrequent = (1 - model_[0].frequent_probability) *
                                embedding_params_.slot_size_array.size() * get_batch_size(true);
  size_t avg_evaluate_infrequent = (1 - model_[0].frequent_probability) *
                                   embedding_params_.slot_size_array.size() * get_batch_size(false);

  HCTR_LOG_S(INFO, ROOT) << "Estimated number of infrequent categories per train batch: "
                         << avg_train_infrequent << ", eval batch: " << avg_evaluate_infrequent
                         << std::endl;

  if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
      (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
    size_t wgrad_size =
        model_[0].num_frequent * embedding_params_.embedding_vec_size * sizeof(emtype);

    if (!grouped_all_reduce_) {
      // Manage your own all-reduce
      auto ar_comm = resource_manager_->get_ar_comm();
      ar_comm->update_size(frequent_embedding_handle_, wgrad_size);
    } else {
      wgrad_offset_in_bytes += wgrad_size;
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::setup_async_mode(AsyncReader<dtype> *train_data_reader,
                                                            AsyncReader<dtype> *eval_data_reader,
                                                            bool eval_overlap,
                                                            bool use_cuda_graph) {
  auto create_async_indices = [this](AsyncReader<dtype> *data_reader, bool is_train) {
    size_t batch_size = get_batch_size(is_train);
    size_t label_dim, dense_dim, sparse_dim, sample_size_items;
    data_reader->get_dimensions(label_dim, dense_dim, sparse_dim, sample_size_items);

    std::vector<FrequentEmbeddingBase<dtype> *> frequent_base_ptrs;
    for (auto &freq : frequent_embeddings_single_node_) {
      frequent_base_ptrs.push_back(dynamic_cast<FrequentEmbeddingBase<dtype> *>(&freq));
    }
    for (auto &freq : frequent_embeddings_multi_node_) {
      frequent_base_ptrs.push_back(dynamic_cast<FrequentEmbeddingBase<dtype> *>(&freq));
    }

    std::vector<InfrequentEmbeddingBase<dtype> *> infrequent_base_ptrs;

    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      for (auto &infreq : infrequent_embeddings_single_node_) {
        infrequent_base_ptrs.push_back(dynamic_cast<InfrequentEmbeddingBase<dtype> *>(&infreq));
      }
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
      for (auto &infreq : infrequent_embeddings_ib_nvlink_) {
        infrequent_base_ptrs.push_back(dynamic_cast<InfrequentEmbeddingBase<dtype> *>(&infreq));
      }
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
      for (auto &infreq : infrequent_embeddings_ib_nvlink_hier_) {
        infrequent_base_ptrs.push_back(dynamic_cast<InfrequentEmbeddingBase<dtype> *>(&infreq));
      }
    }

    return std::make_shared<IndexProcessor<dtype>>(
        model_, frequent_base_ptrs, infrequent_base_ptrs, resource_manager_,
        // double buffer for train, cache each batch for eval
        is_train ? 2 : data_reader->get_total_queue_size(), batch_size,
        embedding_params_.slot_size_array, embedding_params_.max_num_frequent_categories,
        data_reader->is_mixed_precision(), embedding_params_.communication_type, label_dim,
        dense_dim, sparse_dim, sample_size_items);
  };

  if (embedding_params_.use_train_precompute_indices) {
    train_async_indices_ = create_async_indices(train_data_reader, true);
    train_data_reader->register_extra_processing(train_async_indices_, false, use_cuda_graph);
  }
  if (embedding_params_.use_eval_precompute_indices) {
    eval_async_indices_ = create_async_indices(eval_data_reader, false);
    eval_data_reader->register_extra_processing(eval_async_indices_, eval_overlap, use_cuda_graph);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::index_calculation(bool is_train, bool is_first_batch,
                                                             int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (is_train && embedding_params_.use_train_precompute_indices) {
    // Async indices, need to do nothing at all here
  } else if (!is_train && embedding_params_.use_eval_precompute_indices) {
    // Async indices, need to do nothing at all here
  } else {
    auto frequent_indices = (is_train) ? &frequent_embedding_train_indices_[i]
                                       : &frequent_embedding_evaluate_indices_[i];
    auto infrequent_indices = (is_train) ? &infrequent_embedding_train_indices_[i]
                                         : &infrequent_embedding_evaluate_indices_[i];

    auto data = (is_train) ? &data_train_[i] : &data_evaluate_[i];
    auto input = (is_train) ? train_input_tensors_[i].get_value_tensor()
                            : evaluate_input_tensors_[i].get_value_tensor();

    if (is_first_batch) {
      auto &before_idx_event = stream_manager_.get_event(i, "before_idx");
      auto &set_idx_stream = stream_manager_.get_stream(i, "set_idx_stream");
      HCTR_LIB_THROW(hipEventRecord(before_idx_event, stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(set_idx_stream, before_idx_event));
    }

    data->data_to_unique_categories(input, stream);

    compute_indices(*frequent_indices, *infrequent_indices, embedding_params_.communication_type,
                    is_train || is_first_batch, stream, stream_manager_, i,
                    resource_manager_->get_local_gpu(i)->get_sm_count());

    // Setting the indices involves hipMemcpy, so we'll only do that
    // for the first batch after we switch from train to eval (and from eval to train)
    if (is_first_batch) {
      auto &set_idx_stream = stream_manager_.get_stream(i, "set_idx_stream");
      auto &set_idx_event = stream_manager_.get_event(i, "set_idx");

      get_frequent_embedding(i).set_current_indices(frequent_indices, stream);

      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        infrequent_embeddings_single_node_[i].set_current_indices(infrequent_indices, stream);
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        infrequent_embeddings_ib_nvlink_[i].set_current_indices(infrequent_indices, stream);
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_[i].set_current_indices(infrequent_indices, stream);
      }

      HCTR_LIB_THROW(hipEventRecord(set_idx_event, set_idx_stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, set_idx_event));
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train, bool is_first_batch, int i,
                                                   hipStream_t stream, hipEvent_t *evt_ptr) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);

  auto &output = (is_train) ? train_output_tensors_[i] : evaluate_output_tensors_[i];

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    //// TODO: These copies need to be moved to the index computation
    // TODO, need to split into two parts? before and after frequent_emebedding
    infrequent_embeddings_ib_nvlink_[i].forward(output.get_ptr(), stream);

    frequent_embeddings_multi_node_[i].forward_network(output.get_ptr(), stream);

    evt_ptr = nullptr;

  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].forward_model(stream);
    // Let's initiate the communication as soon as we can and start every other non-urgent work
    // here This is for network
    if (is_train) {
      HCTR_LIB_THROW(hipEventRecord(*evt_ptr, stream));
    }

    // This is for frequent forward network running in a side stream
    auto &stream_side = stream_manager_.get_stream(i, "stream_side");
    auto &ready_freq_fwd_net = stream_manager_.get_event(i, "ready_freq_fwd_net");
    auto &freq_fwd_net_completion = stream_manager_.get_event(i, "freq_fwd_net_completion");

    if (is_train) {
      HCTR_LIB_THROW(hipEventRecord(ready_freq_fwd_net, stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream_side, ready_freq_fwd_net));
    }

    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_->wait_completion(stream);

    if (!is_train) {
      if (eval_overlap_) {
        HCTR_LIB_THROW(hipStreamWaitEvent(stream, gpu.get_event("eval_comm_wait")));
      }
      HCTR_LIB_THROW(hipEventRecord(ready_freq_fwd_net, stream));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream_side, ready_freq_fwd_net));
    }

    frequent_embeddings_multi_node_[i].forward_network(output.get_ptr(), stream_side);

    infrequent_embeddings_ib_nvlink_hier_[i].hier_forward_network(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
        output.get_ptr(), stream);

    // join back frequent forward network
    HCTR_LIB_THROW(hipEventRecord(freq_fwd_net_completion, stream_side));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream, freq_fwd_net_completion));

    if (!is_train) {
      if (eval_overlap_) {
        HCTR_LIB_THROW(hipEventRecord(gpu.get_event("eval_comp_wait"), stream));
      }

      // Global barrier
      HCTR_LIB_THROW(ncclAllReduce((const void *)d_barrier_store_[i].get_ptr(),
                                   d_barrier_store_[i].get_ptr(), sizeof(uint32_t),
                                   NcclDataType<uint32_t>::getType(), ncclSum,
                                   get_local_gpu(i).get_nccl(), stream));
    }
  } else {  // Assuming single node

    infrequent_embeddings_single_node_[i].forward_network_direct(is_train, stream);

    // we just need to update frequent cache once in eval
    if (is_train) {
      frequent_embeddings_single_node_[i].forward_model(stream);
    } else {
      if (is_first_batch) {
        frequent_embeddings_single_node_[i].forward_model_eval(stream);
      }
    }

    // This barrier is needed for two reasons:
    // - Ensure all infrequent vectors have been pushed before mlp
    // - Ensure all frequent vectors have been pushed before forward_network
    gpu_barrier_->sync_all_gpus(stream, i);

    frequent_embeddings_single_node_[i].forward_network(output.get_ptr(), stream);
    evt_ptr = nullptr;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train, bool is_first_batch) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

// Index calculations
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto &gpu = get_local_gpu(i);
    CudaDeviceContext context(gpu.get_device_id());
    auto stream = is_train || !eval_overlap_ ? gpu.get_stream() : gpu.get_stream("eval_comms", -1);
    index_calculation(is_train, is_first_batch, i, stream);
    forward(is_train, is_first_batch, i, stream, nullptr);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::frequent_local_reduce(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (frequent_embeddings_single_node_.size()) {
    frequent_embeddings_single_node_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream);
  } else {
    frequent_embeddings_multi_node_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_pre_communication(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_embeddings_ib_nvlink_[i]
            .infrequent_backward_comm_buffers_->send_buffer.get_ptr(),
        stream);
  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_->update_sizes(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].fused_intra_update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_backward_comm_buffers_->send_buffer_ptrs.get_ptr(),
        stream);
  }
}

// Everything that involves network and can be better overlapped with compute
template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_communications(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
    // Synchronize all GPUs before pulling the reduced gradients
    gpu_barrier_->sync_all_gpus(stream, i);

    float *dev_lr = lr_scheds_[i]->get_learning_rate();
    float scale = opt_params_[i].scaler;
    frequent_embeddings_single_node_[i].update_model_direct(dev_lr, scale, stream);

    infrequent_embeddings_single_node_[i].update_model_direct(dev_lr, scale, stream);
  } else {
    if (!grouped_all_reduce_) {
      frequent_embeddings_multi_node_[i].communicate(stream);
    }

    if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
      infrequent_embeddings_ib_nvlink_[i].infrequent_backward_comms_->communicate(stream);
    } else {  // IB_NVLink_Hier
      infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_->communicate(stream);
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::frequent_update(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type != CommunicationType::NVLink_SingleNode) {
    frequent_embeddings_multi_node_[i].update_model(dev_lr, scale, stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_post_communication(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].update_model(
        infrequent_embeddings_ib_nvlink_[i]
            .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
        dev_lr, scale, stream);
  }
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI

    infrequent_embeddings_ib_nvlink_hier_[i].hier_update_model(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
        dev_lr, scale, stream);

    if (graph_mode_) {
      hipEvent_t update_comm_event = stream_manager_.get_event(i, "update_comm_event");
      HCTR_LIB_THROW(hipEventRecord(update_comm_event, comm_stream_[i]));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, update_comm_event));
    }

#else
    HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto stream = get_local_gpu(i).get_stream();
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);
    frequent_local_reduce(i, stream);
    backward_pre_communication(i, stream);
    backward_communications(i, stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::update_params() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto stream = get_local_gpu(i).get_stream();
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);
    frequent_update(i, stream);
    backward_post_communication(i, stream);
  }
}

template <typename dtype, typename emtype>
TrainState HybridSparseEmbedding<dtype, emtype>::train(bool is_train, int i, TrainState state) {
  auto &stream = stream_manager_.get_stream(i, "main_stream");
  auto &ready_bot_mlp_fprop = stream_manager_.get_event(i, "ready_bot_mlp_fprop");
  auto &ready_top_mlp_fprop = stream_manager_.get_event(i, "ready_top_mlp_fprop");
  auto &finish_backward_pre = stream_manager_.get_event(i, "finish_backward_pre");
  auto &finish_iteration = stream_manager_.get_event(i, "finish_iteration");

  auto sync = [&state, &stream]() {
    if (state.event) {
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, *state.event));
    }
  };

  hipEvent_t *event_ptr = nullptr;
  switch (state.state) {
    case TrainState_t::Init:
      sync();
      index_calculation(is_train, -1, i, stream);
      forward(is_train, -1, i, stream, &ready_bot_mlp_fprop);
      event_ptr = &ready_bot_mlp_fprop;
      break;
    case TrainState_t::BottomMLPFprop:
      sync();
      break;
    case TrainState_t::TopMLPFprop:
      HCTR_LIB_THROW(hipEventRecord(ready_top_mlp_fprop, stream));
      event_ptr = &ready_top_mlp_fprop;
      break;
    case TrainState_t::TopMLPBprop:
      break;
    case TrainState_t::BottomMLPBprop:
      if (overlap_ar_a2a_) {
        sync();
        frequent_local_reduce(i, stream);
      }
      break;
    case TrainState_t::MLPExchangeWgrad:
      if (!overlap_ar_a2a_) {
        sync();
        frequent_local_reduce(i, stream);
        backward_pre_communication(i, stream);
      }
      if (grouped_all_reduce_) {
        HCTR_LIB_THROW(hipEventRecord(finish_backward_pre, stream));
        event_ptr = &finish_backward_pre;
      }
      if (overlap_ar_a2a_) {
        backward_pre_communication(i, stream);
        backward_communications(i, stream);
        backward_post_communication(i, stream);
      }
      break;
    case TrainState_t::MLPUpdate:
      if (!overlap_ar_a2a_) {
        sync();
        backward_communications(i, stream);
        frequent_update(i, stream);
        backward_post_communication(i, stream);
      } else {
        sync();
        frequent_update(i, stream);
      }
      break;
    case TrainState_t::Finalize:
      HCTR_LIB_THROW(hipEventRecord(finish_iteration, stream));
      event_ptr = &finish_iteration;
      break;
    default:
      HCTR_OWN_THROW(Error_t::InvalidEnv, "hybrid embedding train reach invalid status");
  }
  state.event = event_ptr;
  return state;
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_params() {
  // TODO: create init_params()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::load_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) {
  // TODO: create load_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::dump_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) const {
  // TODO: create dump_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::set_learning_rate(float lr) {
  HCTR_OWN_THROW(Error_t::WrongInput, "HybridSparseEmbedding only supports GPU LR scheduler");
}

template <typename dtype, typename emtype>
GpuLearningRateSchedulers HybridSparseEmbedding<dtype, emtype>::get_learning_rate_schedulers()
    const {
  return lr_scheds_;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_params_num() const {
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_vocabulary_size() const {
  // TODO: create get_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_max_vocabulary_size() const {
  // TODO: create get_max_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_train_output_tensors() const {
  return tensors_to_bags(train_output_tensors_);
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_evaluate_output_tensors() const {
  return tensors_to_bags(evaluate_output_tensors_);
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::compute_indices(
    FrequentEmbeddingCompression<dtype> &compression,
    InfrequentEmbeddingSelection<dtype> &selection, CommunicationType communication_type,
    bool compute_network_cache_indices, hipStream_t main_stream, StreamManager &manager,
    int raw_device_id, int sm_count) {
  hipStream_t stream_frequent_sample_indices =
      manager.get_stream(raw_device_id, "stream_frequent_sample_indices");
  hipStream_t stream_model_indices = manager.get_stream(raw_device_id, "stream_model_indices");
  hipStream_t stream_network_indices = manager.get_stream(raw_device_id, "stream_network_indices");

  hipEvent_t event_main = manager.get_event(raw_device_id, "event_main");
  hipEvent_t event_frequent_sample_indices =
      manager.get_event(raw_device_id, "event_frequent_sample_indices");
  hipEvent_t event_model_indices = manager.get_event(raw_device_id, "event_model_indices");
  hipEvent_t event_network_indices = manager.get_event(raw_device_id, "event_network_indices");

  // The new streams can only start after previous work in the main stream has completed
  HCTR_LIB_THROW(hipEventRecord(event_main, main_stream));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_frequent_sample_indices, event_main));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_model_indices, event_main));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_network_indices, event_main));

  compression.calculate_frequent_sample_indices(stream_frequent_sample_indices);
  HCTR_LIB_THROW(hipEventRecord(event_frequent_sample_indices, stream_frequent_sample_indices));

  selection.calculate_model_indices(stream_model_indices);
  HCTR_LIB_THROW(hipEventRecord(event_model_indices, stream_model_indices));

  if (communication_type != CommunicationType::NVLink_SingleNode) {
    selection.calculate_network_indices(sm_count, stream_network_indices);
    HCTR_LIB_THROW(hipEventRecord(event_network_indices, stream_network_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_network_indices));

  } else {
    hipStream_t stream_cache_masks = manager.get_stream(raw_device_id, "stream_cache_masks");
    hipStream_t stream_network_cache_indices =
        manager.get_stream(raw_device_id, "stream_network_cache_indices");
    hipStream_t stream_model_cache_indices =
        manager.get_stream(raw_device_id, "stream_model_cache_indices");
    hipEvent_t event_cache_masks = manager.get_event(raw_device_id, "event_cache_masks");
    hipEvent_t event_network_cache_indices =
        manager.get_event(raw_device_id, "event_network_cache_indices");
    hipEvent_t event_model_cache_indices =
        manager.get_event(raw_device_id, "event_model_cache_indices");

    HCTR_LIB_THROW(hipStreamWaitEvent(stream_cache_masks, event_main));

    compression.calculate_cache_masks(stream_cache_masks);
    HCTR_LIB_THROW(hipEventRecord(event_cache_masks, stream_cache_masks));

    HCTR_LIB_THROW(hipStreamWaitEvent(stream_network_cache_indices, event_cache_masks));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream_model_cache_indices, event_cache_masks));

    // we don't need to calculate cache indices during eval
    if (compute_network_cache_indices) {
      compression.calculate_network_cache_indices(stream_network_cache_indices);
    }
    HCTR_LIB_THROW(hipEventRecord(event_network_cache_indices, stream_network_cache_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_network_cache_indices));

    compression.calculate_model_cache_indices(sm_count, stream_model_cache_indices);
    HCTR_LIB_THROW(hipEventRecord(event_model_cache_indices, stream_model_cache_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_model_cache_indices));
  }

  // Join streams to the main stream
  HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_frequent_sample_indices));
  HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_model_indices));
}

template class HybridSparseEmbedding<uint32_t, __half>;
template class HybridSparseEmbedding<uint32_t, float>;
template class HybridSparseEmbedding<long long, __half>;
template class HybridSparseEmbedding<long long, float>;
}  // namespace HugeCTR
