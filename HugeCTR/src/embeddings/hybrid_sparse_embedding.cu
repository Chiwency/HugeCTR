#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <collectives/all_reduce_comm.hpp>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/calibration_data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/indices_container.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/statistics.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/embeddings/hybrid_sparse_embedding.hpp"
#include "HugeCTR/include/tensor2.hpp"

namespace HugeCTR {
template <typename dtype, typename emtype>
HybridSparseEmbedding<dtype, emtype>::HybridSparseEmbedding(
    const SparseTensors<dtype> &train_input_tensors,
    const SparseTensors<dtype> &evaluate_input_tensors,
    const HybridSparseEmbeddingParams &embedding_params,
    const std::vector<BuffPtr<emtype>> &grouped_wgrad_buff,
    const GpuLearningRateSchedulers lr_scheds, bool graph_mode,
    const std::shared_ptr<ResourceManager> &resource_manager, bool overlap_ar_a2a,
    bool eval_overlap)
    : embedding_params_(embedding_params),
      resource_manager_(resource_manager),
      stream_manager_(resource_manager->get_local_gpu_count()),
      grouped_wgrad_buff_(grouped_wgrad_buff),
      grouped_all_reduce_(grouped_wgrad_buff[0] != NULL),
      lr_scheds_(lr_scheds),
      graph_mode_(graph_mode),
      overlap_ar_a2a_(overlap_ar_a2a),
      eval_overlap_(eval_overlap),
      current_train_batch_size_(get_batch_size(true)),
      current_eval_batch_size_(get_batch_size(false)),
      use_graph_(graph_mode) {
  try {
    // 0. Error check
    if (embedding_params_.train_batch_size < 1 || embedding_params_.evaluate_batch_size < 1 ||
        embedding_params_.slot_num < 1 || embedding_params_.embedding_vec_size < 1) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "batchsize < 1 || slot_num < 1 || embedding_vec_size < 1");
    }

    if (embedding_params_.embedding_vec_size > 1024) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "the embedding_vec_size can not be more than 1024 in embedding layer");
    }

    size_t total_gpu_count = resource_manager_->get_global_gpu_count();
    size_t local_gpu_count = resource_manager_->get_local_gpu_count();

    if (train_input_tensors.size() != local_gpu_count ||
        evaluate_input_tensors.size() != local_gpu_count) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "either train_input_tensors.size() or evaluate_input_tensors.size() isn't "
                     "local_gpu_count_");
    }

    HCTR_LOG_S(INFO, ROOT) << "Using Hybrid Embedding with train batch " << get_batch_size(true)
                           << " and eval batch " << get_batch_size(false) << std::endl;

    // 1. initialize optimizer
    for (size_t id = 0; id < local_gpu_count; id++) {
      OptParams opt_params;
      opt_params.optimizer = embedding_params_.opt_params.optimizer;
      opt_params.lr = embedding_params_.opt_params.lr;
      opt_params.update_type = embedding_params_.opt_params.update_type;
      opt_params.scaler = embedding_params_.opt_params.scaler;
      opt_params_.emplace_back(opt_params);
    }
    // 2. reserve buffers for different tensors
    data_statistics_.reserve(local_gpu_count);
    model_.reserve(local_gpu_count);
    calibration_.reserve(local_gpu_count);
    statistics_.reserve(local_gpu_count);
    train_output_tensors_.reserve(local_gpu_count);
    evaluate_output_tensors_.reserve(local_gpu_count);
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      frequent_embeddings_single_node_.reserve(local_gpu_count);
    } else {
      frequent_embeddings_multi_node_.reserve(local_gpu_count);
    }

    infrequent_embeddings_single_node_.reserve(local_gpu_count);
    infrequent_embeddings_ib_nvlink_.reserve(local_gpu_count);
    infrequent_embeddings_ib_nvlink_hier_.reserve(local_gpu_count);

    assert(bufs_.empty());
    CudaDeviceContext context;
    // 2.1. construct data
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      data_statistics_.emplace_back(embedding_params_.slot_size_array, get_batch_size(true),
                                    embedding_params_.num_iterations_statistics);
    }

    // 2.2 construct model
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);

      std::vector<uint32_t> num_instances_per_node(resource_manager_->get_num_process(), 0);
      get_num_instances_per_node(num_instances_per_node);
      model_.emplace_back(embedding_params_.communication_type,
                          resource_manager_->get_local_gpu(i)->get_global_id(),
                          num_instances_per_node, get_categories_num());
    }

    // 2.3 construct calibration
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      calibration_.emplace_back(resource_manager_->get_num_process(), embedding_params_.p_dup_max,
                                embedding_params_.max_all_reduce_bandwidth,
                                embedding_params_.max_all_to_all_bandwidth,
                                embedding_params_.efficiency_bandwidth_ratio);
    }

    // 2.4 construct Statistics
    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      const size_t num_samples_statistics = embedding_params_.num_iterations_statistics *
                                            get_batch_size(true) * embedding_params_.slot_num;
      statistics_.emplace_back((dtype)num_samples_statistics, embedding_params_.slot_num,
                               model_[i].num_instances, get_categories_num());
    }

    for (uint32_t i = 0; i < local_gpu_count; i++) {
      int cur_device = get_local_gpu(i).get_device_id();
      context.set_device(cur_device);
      std::shared_ptr<GeneralBuffer2<CudaAllocator>> buf = GeneralBuffer2<CudaAllocator>::create();
      bufs_.emplace_back(buf);
      // 2.5. reserve for train output/ evaluate output tensors
      Tensor2<emtype> tensor;
      buf->reserve({get_batch_size_per_gpu(true), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      train_output_tensors_.emplace_back(tensor);
      buf->reserve({get_batch_size_per_gpu(false), get_slot_num(), get_embedding_vec_size()},
                   &tensor);
      evaluate_output_tensors_.emplace_back(tensor);

      // 2.6 construct frequent embedding
      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        frequent_embeddings_single_node_.emplace_back(
            model_[i], get_local_gpu(i), grouped_wgrad_buff_[i], get_embedding_vec_size(),
            embedding_params_.max_num_frequent_categories);
      } else {
        frequent_embeddings_multi_node_.emplace_back(
            model_[i], get_local_gpu(i), grouped_wgrad_buff_[i], get_embedding_vec_size(),
            embedding_params_.max_num_frequent_categories);
      }

      // 2.7 construct infrequent embedding
      if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
        infrequent_embeddings_single_node_.emplace_back(model_[i], get_local_gpu(i),
                                                        get_embedding_vec_size());
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        infrequent_embeddings_ib_nvlink_.emplace_back(model_[i], get_local_gpu(i),
                                                      get_embedding_vec_size());
      }
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_.emplace_back(model_[i], get_local_gpu(i),
                                                           get_embedding_vec_size());
      }

      // 2.8 construct communication
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
        size_t max_buf_size = embedding_params_.embedding_vec_size *
                              std::max(get_batch_size(true), get_batch_size(false)) *
                              embedding_params_.slot_num;
        infrequent_embeddings_ib_nvlink_.back().init_comms(
            embedding_params_.embedding_vec_size, &get_local_gpu(i), buf.get(), max_buf_size);
      }

      // Construct comm buffers
      if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
        infrequent_embeddings_ib_nvlink_hier_[i].init_comms(
            embedding_params_.max_num_infrequent_samples, embedding_params_.slot_num,
            embedding_params_.embedding_vec_size, buf.get(), get_batch_size(true),
            get_batch_size(false), local_gpu_count);
      }

      // For global barrier in eval
      {
        Tensor2<uint32_t> tensor;
        buf->reserve({1}, &tensor);
        d_barrier_store_.push_back(tensor);
      }
      buf->allocate();
    }

    // Frequent AR comm init
    if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
        (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
      if (!grouped_all_reduce_) {
        // Do your own all-reduce
        auto ar_comm = resource_manager_->get_ar_comm();
        frequent_embedding_handle_ = ar_comm->register_coll();
        // Frequent all reduce comm
        for (uint32_t i = 0; i < local_gpu_count; i++) {
          frequent_embeddings_multi_node_[i].init_ar_comm(ar_comm, frequent_embedding_handle_, i);
        }
        ar_comm->register_coll_buf(frequent_embedding_handle_);
      }
    }

    // Init after buffer allocation
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI
      ib_comm_ = resource_manager_->get_ib_comm();
      if (!ib_comm_) {
        resource_manager_->init_ib_comm();
        ib_comm_ = resource_manager_->get_ib_comm();
      }
      comm_stream_.resize(local_gpu_count);

      std::vector<size_t *> h_model_indices_sizes_ptrs(local_gpu_count);
      std::vector<size_t *> h_network_indices_sizes_ptrs(local_gpu_count);
      std::vector<emtype *> h_fwd_send_buffer_ptrs(local_gpu_count);
      std::vector<emtype *> h_bwd_send_buffer_ptrs(local_gpu_count);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        h_model_indices_sizes_ptrs[i] =
            infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_.get_ptr();
        h_network_indices_sizes_ptrs[i] =
            infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_.get_ptr();
        h_fwd_send_buffer_ptrs[i] = infrequent_embeddings_ib_nvlink_hier_[i]
                                        .infrequent_forward_comm_buffers_->send_buffer.get_ptr();
        h_bwd_send_buffer_ptrs[i] = infrequent_embeddings_ib_nvlink_hier_[i]
                                        .infrequent_backward_comm_buffers_->send_buffer.get_ptr();
      }

      // Forward coll init
      auto infrequent_forward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        // download pointers
        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_ptrs_.get_ptr(),
            h_model_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipMemcpyAsync(
            infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_ptrs_.get_ptr(),
            h_network_indices_sizes_ptrs.data(), sizeof(size_t *) * local_gpu_count,
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_ib_nvlink_hier_[i]
                                .infrequent_forward_comm_buffers_->send_buffer_ptrs.get_ptr(),
                            h_fwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(
            hipMemcpyAsync(infrequent_embeddings_ib_nvlink_hier_[i]
                                .infrequent_backward_comm_buffers_->send_buffer_ptrs.get_ptr(),
                            h_bwd_send_buffer_ptrs.data(), sizeof(emtype *) * local_gpu_count,
                            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        HCTR_LIB_THROW(hipStreamSynchronize(get_local_gpu(i).get_stream()));

        // Initialize IB comm
        HCTR_LIB_THROW(hipStreamCreateWithPriority(&comm_stream_[i], hipStreamNonBlocking, -100));
        ib_comm_->set_a2a_coll_stream(infrequent_forward_coll_handle, comm_stream_[i], i);

        ib_comm_->set_a2a_coll_buf(
            infrequent_forward_coll_handle,
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->send_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->send_buffer.get_size_in_bytes(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_forward_comm_buffers_->recv_buffer.get_size_in_bytes(),
            i);

        infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_ =
            std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
                i, infrequent_forward_coll_handle,
                infrequent_embeddings_ib_nvlink_hier_[i].model_indices_sizes_ptrs_.get_ptr(),
                &get_local_gpu(i), ib_comm_, comm_stream_[i]);
      }
      ib_comm_->register_a2a_coll_buf(infrequent_forward_coll_handle);

      // Backward coll init
      auto infrequent_backward_coll_handle = ib_comm_->register_hier_a2a_v_coll(true);
      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        ib_comm_->set_a2a_coll_stream(infrequent_backward_coll_handle, comm_stream_[i], i);
        ib_comm_->set_a2a_coll_buf(
            infrequent_backward_coll_handle,
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->send_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->send_buffer.get_size_in_bytes(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
            infrequent_embeddings_ib_nvlink_hier_[i]
                .infrequent_backward_comm_buffers_->recv_buffer.get_size_in_bytes(),
            i);

        infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_ =
            std::make_unique<HierAll2Allv_Multi_IB<emtype>>(
                i, infrequent_backward_coll_handle,
                infrequent_embeddings_ib_nvlink_hier_[i].network_indices_sizes_ptrs_.get_ptr(),
                &get_local_gpu(i), ib_comm_, comm_stream_[i]);
      }
      ib_comm_->register_a2a_coll_buf(infrequent_backward_coll_handle);
#else
      HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
    }

    // 2.9 Single-node: copy some pointers arrays to device
    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      // Initialize GPU barrier
      gpu_barrier_ = std::make_unique<GPUBarrier>(resource_manager_->get_local_gpu_count(),
                                                  resource_manager_->get_local_gpu_device_id_list(),
                                                  graph_mode_);

      std::vector<const emtype *> frequent_vectors_cache_pointers(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_train(local_gpu_count);
      std::vector<emtype *> interaction_layer_input_pointers_eval(local_gpu_count);
      std::vector<const emtype *> gradients_pointers(local_gpu_count);
      std::vector<const emtype *> frequent_partial_gradients_pointers(local_gpu_count);

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        frequent_vectors_cache_pointers[i] =
            frequent_embeddings_single_node_[i].get_embedding_vectors_cache().get_ptr();
        interaction_layer_input_pointers_train[i] = train_output_tensors_[i].get_ptr();
        gradients_pointers[i] = train_output_tensors_[i].get_ptr();
        interaction_layer_input_pointers_eval[i] = evaluate_output_tensors_[i].get_ptr();
        frequent_partial_gradients_pointers[i] =
            frequent_embeddings_single_node_[i].frequent_data_.get_gradients().get_ptr();
      }

      for (uint32_t i = 0; i < local_gpu_count; i++) {
        int cur_device = get_local_gpu(i).get_device_id();
        context.set_device(cur_device);

        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_single_node_[i].embedding_vectors_cache_pointers_.get_ptr(),
            frequent_vectors_cache_pointers.data(), local_gpu_count * sizeof(float *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));

        infrequent_embeddings_single_node_[i].init_pointers(
            local_gpu_count, get_local_gpu(i).get_stream(), interaction_layer_input_pointers_train,
            interaction_layer_input_pointers_eval, gradients_pointers);
        HCTR_LIB_THROW(hipMemcpyAsync(
            frequent_embeddings_single_node_[i].partial_gradients_pointers_.get_ptr(),
            frequent_partial_gradients_pointers.data(), local_gpu_count * sizeof(emtype *),
            hipMemcpyHostToDevice, get_local_gpu(i).get_stream()));
      }
    }

    // Setup default indices
    train_batch_indices_.emplace_back(model_, train_input_tensors, resource_manager_,
                                      get_batch_size(true), embedding_params_.slot_size_array,
                                      embedding_params_.max_num_frequent_categories,
                                      embedding_params_.communication_type);

    eval_batch_indices_.emplace_back(model_, evaluate_input_tensors, resource_manager_,
                                     get_batch_size(false), embedding_params_.slot_size_array,
                                     embedding_params_.max_num_frequent_categories,
                                     embedding_params_.communication_type);

  } catch (const std::runtime_error &rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_model(const SparseTensors<dtype> &data,
                                                      size_t &wgrad_offset_in_bytes) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();
  HCTR_LOG(INFO, ROOT, "Initializing Hybrid Embedding\n");
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t id = 0; id < local_gpu_count; ++id) {
    int cur_device = get_local_gpu(id).get_device_id();
    CudaDeviceContext context(cur_device);
    auto stream = get_local_gpu(id).get_stream();
    data_statistics_[id].data_to_unique_categories(data[id].get_value_tensor(), stream);
    model_[id].init_hybrid_model(calibration_[id], statistics_[id], data_statistics_[id], stream);

    get_frequent_embedding_data(id).initialize_embedding_vectors(data_statistics_[id].table_sizes,
                                                                 wgrad_offset_in_bytes);

    if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
      infrequent_embeddings_single_node_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
      infrequent_embeddings_ib_nvlink_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }
    if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
      infrequent_embeddings_ib_nvlink_hier_[id].initialize_embedding_vectors(
          data_statistics_[id].table_sizes);
    }

    if (embedding_params_.max_num_frequent_categories < (size_t)model_[id].num_frequent) {
      HCTR_OWN_THROW(
          Error_t::WrongInput,
          "Found too many frequent categories, please increase 'max_num_frequent_categories'");
    }
  }

  HCTR_LOG_S(INFO, ROOT) << "Initialized hybrid model with " << model_[0].num_frequent
                         << " frequent categories, probability of being frequent is "
                         << model_[0].frequent_probability << std::endl;

  size_t avg_train_infrequent = (1 - model_[0].frequent_probability) *
                                embedding_params_.slot_size_array.size() * get_batch_size(true);
  size_t avg_evaluate_infrequent = (1 - model_[0].frequent_probability) *
                                   embedding_params_.slot_size_array.size() * get_batch_size(false);

  HCTR_LOG_S(INFO, ROOT) << "Estimated number of infrequent categories per train batch: "
                         << avg_train_infrequent << ", eval batch: " << avg_evaluate_infrequent
                         << std::endl;

  if ((embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) ||
      (embedding_params_.communication_type == CommunicationType::IB_NVLink)) {
    size_t wgrad_size =
        model_[0].num_frequent * embedding_params_.embedding_vec_size * sizeof(emtype);

    if (!grouped_all_reduce_) {
      // Manage your own all-reduce
      auto ar_comm = resource_manager_->get_ar_comm();
      ar_comm->update_size(frequent_embedding_handle_, wgrad_size);
    } else {
      wgrad_offset_in_bytes += wgrad_size;
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::setup_buffered_indices(
    AsyncReader<dtype> *train_data_reader, AsyncReader<dtype> *eval_data_reader) {
  auto create_buffered_indices = [this](AsyncReader<dtype> *reader, bool is_train) {
    const auto data_tensors = reader->get_value_tensor_buffers();
    auto &batch_indices = is_train ? train_batch_indices_ : eval_batch_indices_;
    batch_indices.clear();  // remove default
    for (size_t i = 0; i < data_tensors.size(); ++i) {
      batch_indices.emplace_back(model_, data_tensors.at(i), resource_manager_,
                                 get_batch_size(is_train), embedding_params_.slot_size_array,
                                 embedding_params_.max_num_frequent_categories,
                                 embedding_params_.communication_type);
    }
    return batch_indices;
  };

  if (embedding_params_.use_eval_precompute_indices) {
    // If get_max_batches_inflight() is > than the number of eval batches in the dataset,
    // this will cause the batch tensors to be cached. We need the tensors to be cached in order
    // for the indices to be cached because the index calculation is done in place in these
    // tensors.
    eval_data_reader->set_tensor_buffering(eval_data_reader->get_max_batches_inflight());
    create_buffered_indices(eval_data_reader, false);
  }

  if (embedding_params_.use_train_precompute_indices) {
    // Double buffering for overlapping indices calculation between iterations
    train_data_reader->set_tensor_buffering(2);
    create_buffered_indices(train_data_reader, true);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::index_calculation(bool is_train, int i,
                                                             hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  auto &batch_indices = is_train ? train_batch_indices_.at(train_inflight_id_)
                                 : eval_batch_indices_.at(eval_inflight_id_);

  if (is_train) {
    if (!current_train_batch_cached_) {
      batch_indices.compute(i, current_train_batch_size_, stream);
    }
  } else {  // eval
    if (!current_eval_batch_cached_) {
      batch_indices.compute(i, current_eval_batch_size_, stream);
    }
  }

  // We don't copy the sparse tensor since all the required data are already in the
  // Data type and indices
  get_frequent_embedding(i).set_current_indices(&batch_indices.get_frequent(i));
  get_infrequent_embedding(i).set_current_indices(&batch_indices.get_infrequent(i));

  // Only overlap index calculation with next iteration. Prevent next steps in training pipeline
  // from overlapping, so we wait for the previous iteration to end before continuing.
  if (is_train) {
    auto &iteration_end = get_local_gpu(i).get_event("iteration_end");
    HCTR_LIB_THROW(
        hipStreamWaitEvent(stream, iteration_end, use_graph_ ? cudaEventWaitExternal : 0));
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train, bool is_first_batch, int i,
                                                   hipStream_t stream, hipEvent_t *evt_ptr) {
  int cur_device = get_local_gpu(i).get_device_id();
  auto &gpu = get_local_gpu(i);
  CudaDeviceContext context(cur_device);

  auto &output = (is_train) ? train_output_tensors_[i] : evaluate_output_tensors_[i];

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    //// TODO: These copies need to be moved to the index computation
    // TODO, need to split into two parts? before and after frequent_emebedding
    infrequent_embeddings_ib_nvlink_[i].forward(output.get_ptr(), stream);

    frequent_embeddings_multi_node_[i].forward_network(output.get_ptr(), stream);

    evt_ptr = nullptr;

  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].forward_model(stream);
    // Let's initiate the communication as soon as we can and start every other non-urgent work
    // here This is for network
    if (is_train) {
      HCTR_LIB_THROW(hipEventRecord(*evt_ptr, stream));
    }

    // This is for frequent forward network running in a side stream
    auto &stream_side = stream_manager_.get_stream(i, "stream_side");
    auto &ready_freq_fwd_net = stream_manager_.get_event(i, "ready_freq_fwd_net");
    auto &freq_fwd_net_completion = stream_manager_.get_event(i, "freq_fwd_net_completion");

    if (!is_train) {
      if (eval_overlap_) {
        HCTR_LIB_THROW(hipStreamWaitEvent(stream, gpu.get_event("eval_comm_wait")));
      }
    }

    HCTR_LIB_THROW(hipEventRecord(ready_freq_fwd_net, stream));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream_side, ready_freq_fwd_net));

    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_forward_comms_->wait_completion(stream);
    frequent_embeddings_multi_node_[i].forward_network(output.get_ptr(), stream_side);

    infrequent_embeddings_ib_nvlink_hier_[i].hier_forward_network(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_forward_comm_buffers_->recv_buffer.get_ptr(),
        output.get_ptr(), stream);

    // join back frequent forward network
    HCTR_LIB_THROW(hipEventRecord(freq_fwd_net_completion, stream_side));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream, freq_fwd_net_completion));

    if (!is_train) {
      if (eval_overlap_) {
        HCTR_LIB_THROW(hipEventRecord(gpu.get_event("eval_comp_wait"), stream));
      }

      // Global barrier
      HCTR_LIB_THROW(ncclAllReduce((const void *)d_barrier_store_[i].get_ptr(),
                                   d_barrier_store_[i].get_ptr(), sizeof(uint32_t),
                                   NcclDataType<uint32_t>::getType(), ncclSum,
                                   get_local_gpu(i).get_nccl(), stream));
    }
  } else {  // Assuming single node

    infrequent_embeddings_single_node_[i].forward_network_direct(is_train, stream);

    // we just need to update frequent cache once in eval
    if (is_train) {
      frequent_embeddings_single_node_[i].forward_model(stream);
    } else {
      if (is_first_batch) {
        frequent_embeddings_single_node_[i].forward_model_eval(stream);
      }
    }

    // This barrier is needed for two reasons:
    // - Ensure all infrequent vectors have been pushed before mlp
    // - Ensure all frequent vectors have been pushed before forward_network
    gpu_barrier_->sync_all_gpus(stream, i);

    frequent_embeddings_single_node_[i].forward_network(output.get_ptr(), stream);
    evt_ptr = nullptr;
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::forward(bool is_train, bool is_first_batch) {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

// Index calculations
#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto &gpu = get_local_gpu(i);
    CudaDeviceContext context(gpu.get_device_id());
    auto stream = is_train || !eval_overlap_ ? gpu.get_stream() : gpu.get_stream("eval_comms", -1);
    index_calculation(is_train, i, stream);
    forward(is_train, is_first_batch, i, stream, nullptr);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::frequent_local_reduce(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (frequent_embeddings_single_node_.size()) {
    frequent_embeddings_single_node_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream);
  } else {
    frequent_embeddings_multi_node_[i].local_reduce(train_output_tensors_[i].get_ptr(), stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_pre_communication(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_embeddings_ib_nvlink_[i]
            .infrequent_backward_comm_buffers_->send_buffer.get_ptr(),
        stream);
  } else if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
    infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_->update_sizes(stream);
    infrequent_embeddings_ib_nvlink_hier_[i].fused_intra_update_network(
        train_output_tensors_[i].get_ptr(),
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_backward_comm_buffers_->send_buffer_ptrs.get_ptr(),
        stream);
  }
}

// Everything that involves network and can be better overlapped with compute
template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_communications(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  if (embedding_params_.communication_type == CommunicationType::NVLink_SingleNode) {
    // Synchronize all GPUs before pulling the reduced gradients
    gpu_barrier_->sync_all_gpus(stream, i);

    float *dev_lr = lr_scheds_[i]->get_learning_rate();
    float scale = opt_params_[i].scaler;
    frequent_embeddings_single_node_[i].update_model_direct(dev_lr, scale, stream);

    infrequent_embeddings_single_node_[i].update_model_direct(dev_lr, scale, stream);
  } else {
    if (!grouped_all_reduce_) {
      frequent_embeddings_multi_node_[i].communicate(stream);
    }

    if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
      infrequent_embeddings_ib_nvlink_[i].infrequent_backward_comms_->communicate(stream);
    } else {  // IB_NVLink_Hier
      infrequent_embeddings_ib_nvlink_hier_[i].infrequent_backward_comms_->communicate(stream);
    }
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::frequent_update(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type != CommunicationType::NVLink_SingleNode) {
    frequent_embeddings_multi_node_[i].update_model(dev_lr, scale, stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward_post_communication(int i, hipStream_t stream) {
  int cur_device = get_local_gpu(i).get_device_id();
  CudaDeviceContext context(cur_device);
  float *dev_lr = lr_scheds_[i]->get_learning_rate();
  float scale = opt_params_[i].scaler;

  if (embedding_params_.communication_type == CommunicationType::IB_NVLink) {
    infrequent_embeddings_ib_nvlink_[i].update_model(
        infrequent_embeddings_ib_nvlink_[i]
            .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
        dev_lr, scale, stream);
  }
  if (embedding_params_.communication_type == CommunicationType::IB_NVLink_Hier) {
#ifdef ENABLE_MPI

    infrequent_embeddings_ib_nvlink_hier_[i].hier_update_model(
        infrequent_embeddings_ib_nvlink_hier_[i]
            .infrequent_backward_comm_buffers_->recv_buffer.get_ptr(),
        dev_lr, scale, stream);

    if (graph_mode_) {
      hipEvent_t update_comm_event = stream_manager_.get_event(i, "update_comm_event");
      HCTR_LIB_THROW(hipEventRecord(update_comm_event, comm_stream_[i]));
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, update_comm_event));
    }

#else
    HCTR_OWN_THROW(Error_t::WrongInput, "MPI is not enabled but trying to use IB_NVLink_Hier");
#endif
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::backward() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto stream = get_local_gpu(i).get_stream();
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);
    frequent_local_reduce(i, stream);
    backward_pre_communication(i, stream);
    backward_communications(i, stream);
  }
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::update_params() {
  size_t local_gpu_count = resource_manager_->get_local_gpu_count();

#pragma omp parallel for num_threads(local_gpu_count)
  for (size_t i = 0; i < local_gpu_count; i++) {
    auto stream = get_local_gpu(i).get_stream();
    auto cur_device = get_local_gpu(i).get_device_id();
    CudaDeviceContext context(cur_device);
    frequent_update(i, stream);
    backward_post_communication(i, stream);
  }
}

template <typename dtype, typename emtype>
TrainState HybridSparseEmbedding<dtype, emtype>::train(bool is_train, int i, TrainState state) {
  auto &stream = stream_manager_.get_stream(i, "main_stream");
  auto &ready_bot_mlp_fprop = stream_manager_.get_event(i, "ready_bot_mlp_fprop");
  auto &ready_top_mlp_fprop = stream_manager_.get_event(i, "ready_top_mlp_fprop");
  auto &finish_backward_pre = stream_manager_.get_event(i, "finish_backward_pre");
  auto &finish_iteration = stream_manager_.get_event(i, "finish_iteration");

  auto sync = [&state, &stream]() {
    if (state.event) {
      HCTR_LIB_THROW(hipStreamWaitEvent(stream, *state.event));
    }
  };

  hipEvent_t *event_ptr = nullptr;
  switch (state.state) {
    case TrainState_t::Init:
      sync();
      index_calculation(is_train, i, stream);
      forward(is_train, -1, i, stream, &ready_bot_mlp_fprop);
      event_ptr = &ready_bot_mlp_fprop;
      break;
    case TrainState_t::BottomMLPFprop:
      sync();
      break;
    case TrainState_t::TopMLPFprop:
      HCTR_LIB_THROW(hipEventRecord(ready_top_mlp_fprop, stream));
      event_ptr = &ready_top_mlp_fprop;
      break;
    case TrainState_t::TopMLPBprop:
      break;
    case TrainState_t::BottomMLPBprop:
      if (overlap_ar_a2a_) {
        sync();
        frequent_local_reduce(i, stream);
      }
      break;
    case TrainState_t::MLPExchangeWgrad:
      if (!overlap_ar_a2a_) {
        sync();
        frequent_local_reduce(i, stream);
        backward_pre_communication(i, stream);
      }
      if (grouped_all_reduce_) {
        HCTR_LIB_THROW(hipEventRecord(finish_backward_pre, stream));
        event_ptr = &finish_backward_pre;
      }
      if (overlap_ar_a2a_) {
        backward_pre_communication(i, stream);
        backward_communications(i, stream);
        backward_post_communication(i, stream);
      }
      break;
    case TrainState_t::MLPUpdate:
      if (!overlap_ar_a2a_) {
        sync();
        backward_communications(i, stream);
        frequent_update(i, stream);
        backward_post_communication(i, stream);
      } else {
        sync();
        frequent_update(i, stream);
      }
      break;
    case TrainState_t::Finalize:
      HCTR_LIB_THROW(hipEventRecord(finish_iteration, stream));
      event_ptr = &finish_iteration;
      break;
    default:
      HCTR_OWN_THROW(Error_t::InvalidEnv, "hybrid embedding train reach invalid status");
  }
  state.event = event_ptr;
  return state;
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::init_params() {
  // TODO: create init_params()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::load_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) {
  // TODO: create load_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::dump_parameters(
    std::string sparse_model, const DataSourceParams &data_source_params) const {
  // TODO: create dump_parameters()
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::set_learning_rate(float lr) {
  HCTR_OWN_THROW(Error_t::WrongInput, "HybridSparseEmbedding only supports GPU LR scheduler");
}

template <typename dtype, typename emtype>
GpuLearningRateSchedulers HybridSparseEmbedding<dtype, emtype>::get_learning_rate_schedulers()
    const {
  return lr_scheds_;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_params_num() const {
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_vocabulary_size() const {
  // TODO: create get_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
size_t HybridSparseEmbedding<dtype, emtype>::get_max_vocabulary_size() const {
  // TODO: create get_max_vocabulary_size()
  return 0;
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_train_output_tensors() const {
  return tensors_to_bags(train_output_tensors_);
}

template <typename dtype, typename emtype>
std::vector<TensorBag2> HybridSparseEmbedding<dtype, emtype>::get_evaluate_output_tensors() const {
  return tensors_to_bags(evaluate_output_tensors_);
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::compute_indices(
    FrequentEmbeddingCompression<dtype> &compression,
    InfrequentEmbeddingSelection<dtype> &selection, CommunicationType communication_type,
    bool compute_network_cache_indices, hipStream_t main_stream, StreamManager &manager,
    int raw_device_id, int sm_count) {
  hipStream_t stream_frequent_sample_indices =
      manager.get_stream(raw_device_id, "stream_frequent_sample_indices");
  hipStream_t stream_model_indices = manager.get_stream(raw_device_id, "stream_model_indices");
  hipStream_t stream_network_indices = manager.get_stream(raw_device_id, "stream_network_indices");

  hipEvent_t event_main = manager.get_event(raw_device_id, "event_main");
  hipEvent_t event_frequent_sample_indices =
      manager.get_event(raw_device_id, "event_frequent_sample_indices");
  hipEvent_t event_model_indices = manager.get_event(raw_device_id, "event_model_indices");
  hipEvent_t event_network_indices = manager.get_event(raw_device_id, "event_network_indices");

  // The new streams can only start after previous work in the main stream has completed
  HCTR_LIB_THROW(hipEventRecord(event_main, main_stream));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_frequent_sample_indices, event_main));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_model_indices, event_main));
  HCTR_LIB_THROW(hipStreamWaitEvent(stream_network_indices, event_main));

  compression.calculate_frequent_sample_indices(stream_frequent_sample_indices);
  HCTR_LIB_THROW(hipEventRecord(event_frequent_sample_indices, stream_frequent_sample_indices));

  selection.calculate_model_indices(stream_model_indices);
  HCTR_LIB_THROW(hipEventRecord(event_model_indices, stream_model_indices));

  if (communication_type != CommunicationType::NVLink_SingleNode) {
    selection.calculate_network_indices(sm_count, stream_network_indices);
    HCTR_LIB_THROW(hipEventRecord(event_network_indices, stream_network_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_network_indices));

  } else {
    hipStream_t stream_cache_masks = manager.get_stream(raw_device_id, "stream_cache_masks");
    hipStream_t stream_network_cache_indices =
        manager.get_stream(raw_device_id, "stream_network_cache_indices");
    hipStream_t stream_model_cache_indices =
        manager.get_stream(raw_device_id, "stream_model_cache_indices");
    hipEvent_t event_cache_masks = manager.get_event(raw_device_id, "event_cache_masks");
    hipEvent_t event_network_cache_indices =
        manager.get_event(raw_device_id, "event_network_cache_indices");
    hipEvent_t event_model_cache_indices =
        manager.get_event(raw_device_id, "event_model_cache_indices");

    HCTR_LIB_THROW(hipStreamWaitEvent(stream_cache_masks, event_main));

    compression.calculate_cache_masks(stream_cache_masks);
    HCTR_LIB_THROW(hipEventRecord(event_cache_masks, stream_cache_masks));

    HCTR_LIB_THROW(hipStreamWaitEvent(stream_network_cache_indices, event_cache_masks));
    HCTR_LIB_THROW(hipStreamWaitEvent(stream_model_cache_indices, event_cache_masks));

    // we don't need to calculate cache indices during eval
    if (compute_network_cache_indices) {
      compression.calculate_network_cache_indices(stream_network_cache_indices);
    }
    HCTR_LIB_THROW(hipEventRecord(event_network_cache_indices, stream_network_cache_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_network_cache_indices));

    compression.calculate_model_cache_indices(sm_count, stream_model_cache_indices);
    HCTR_LIB_THROW(hipEventRecord(event_model_cache_indices, stream_model_cache_indices));
    HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_model_cache_indices));
  }

  // Join streams to the main stream
  HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_frequent_sample_indices));
  HCTR_LIB_THROW(hipStreamWaitEvent(main_stream, event_model_indices));
}

template <typename dtype, typename emtype>
void HybridSparseEmbedding<dtype, emtype>::assign_input_tensors(bool is_train, size_t batch_size,
                                                                size_t inflight_id, bool cached,
                                                                bool use_graph) {
  use_graph_ = use_graph;
  if (is_train) {
    train_inflight_id_ = inflight_id;
    current_train_batch_size_ = batch_size;
    current_train_batch_cached_ = cached;
  } else {
    eval_inflight_id_ = inflight_id;
    current_eval_batch_size_ = batch_size;
    current_eval_batch_cached_ = cached;
  }
}

template class HybridSparseEmbedding<uint32_t, __half>;
template class HybridSparseEmbedding<uint32_t, float>;
template class HybridSparseEmbedding<long long, __half>;
template class HybridSparseEmbedding<long long, float>;
}  // namespace HugeCTR
