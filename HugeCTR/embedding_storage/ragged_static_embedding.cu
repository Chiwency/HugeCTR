#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hiprand/hiprand_kernel.h>

#include <data_simulator.hpp>
#include <embedding/operators/generic_lookup.cuh>
#include <embedding/view.hpp>
#include <embedding_storage/ragged_static_embedding.hpp>
#include <numeric>
#include <utils.cuh>

namespace embedding {

namespace {
template <typename key_t, typename index_t>
__global__ void ragged_static_embedding_table_lookup_kernel(
    const key_t *keys, size_t num_keys, const uint32_t *id_space_offset, size_t num_id_space_offset,
    const int *id_space_list, const int *local_id_space_list, size_t num_local_id_space_list,
    const index_t *emb_table_id_space_offset, float *emb_table, const uint64_t *emb_table_ev_offset,
    const int *local_ev_size_list, float **emb_vec) {
  for (uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < num_keys;
       tid += blockDim.x * gridDim.x) {
    int id_space_idx = bs_upper_bound_sub_one(id_space_offset, num_id_space_offset, tid);
    int id_space = id_space_list[id_space_idx];
    int local_id_space_idx =
        bs_upper_bound_sub_one(local_id_space_list, num_local_id_space_list, id_space);

    index_t start = emb_table_id_space_offset[local_id_space_idx];
    uint64_t ev_offset = emb_table_ev_offset[local_id_space_idx];
    int ev_size = local_ev_size_list[local_id_space_idx];

    emb_vec[tid] = &emb_table[ev_offset + static_cast<uint64_t>(keys[tid] - start) * ev_size];
  }
}

template <typename key_t, typename index_t>
struct RaggedKeyToIndicesFunc {
  int *local_table_ids;
  int *local_ev_sizes;
  int64_t num_local_table_ids;

  index_t *emb_table_id_space_offset;
  uint64_t *emb_table_ev_start_indices;

  DEVICE_INLINE void operator()(const key_t &key, const int &table_id,
                                uint64_t *ev_start_indices_ptr, int *ev_size_ptr) {
    int local_id_space_idx = bs_upper_bound_sub_one(local_table_ids, num_local_table_ids, table_id);
    assert(local_id_space_idx >= 0);
    assert(local_id_space_idx < num_local_table_ids);
    index_t start = emb_table_id_space_offset[local_id_space_idx];

    uint64_t ev_offset = emb_table_ev_start_indices[local_id_space_idx];
    int ev_size = local_ev_sizes[local_id_space_idx];

    *ev_start_indices_ptr = ev_offset + static_cast<uint64_t>(key - start) * ev_size;
    *ev_size_ptr = ev_size;
  }
};

template <typename wgrad_t>
struct OptimizierInput {
  const wgrad_t *wgrad;
  uint64_t ev_start_indices;
  int ev_id;
  float lr;
  float scaler;
};

constexpr int num_load_floats = 4;
template <typename wgrad_t>
struct SGDOptimizer {
  DEVICE_INLINE float4 update4(const OptimizierInput<wgrad_t> &input) {
    Vec4T<float> gi;
    gi.load(input.wgrad + input.ev_id, 4);
    float4 ret;
    ret.x = -input.lr * gi.val.x / input.scaler;
    ret.y = -input.lr * gi.val.y / input.scaler;
    ret.z = -input.lr * gi.val.z / input.scaler;
    ret.w = -input.lr * gi.val.w / input.scaler;
    return ret;
  }

  DEVICE_INLINE float update(const OptimizierInput<wgrad_t> &input) {
    return -input.lr *
           (HugeCTR::TypeConvertFunc<float, wgrad_t>::convert(input.wgrad[input.ev_id]) /
            input.scaler);
  }
};

template <typename wgrad_t, typename acc_t>
struct AdaGradOptimizer {
  acc_t *v;
  float epsilon;

  DEVICE_INLINE float4 update4(const OptimizierInput<wgrad_t> &input) {
    Vec4T<float> vi;
    vi.load(v + input.ev_start_indices + input.ev_id, 4);
    Vec4T<float> gi;
    gi.load(input.wgrad + input.ev_id, 4);

    gi.val.x = gi.val.x / input.scaler;
    gi.val.y = gi.val.y / input.scaler;
    gi.val.z = gi.val.z / input.scaler;
    gi.val.w = gi.val.w / input.scaler;
    vi.val.x = vi.val.x + gi.val.x * gi.val.x;
    vi.val.y = vi.val.y + gi.val.y * gi.val.y;
    vi.val.z = vi.val.z + gi.val.z * gi.val.z;
    vi.val.w = vi.val.w + gi.val.w * gi.val.w;

    gi.val.x = -input.lr * gi.val.x / (sqrtf(vi.val.x) + epsilon);
    gi.val.y = -input.lr * gi.val.y / (sqrtf(vi.val.y) + epsilon);
    gi.val.z = -input.lr * gi.val.z / (sqrtf(vi.val.z) + epsilon);
    gi.val.w = -input.lr * gi.val.w / (sqrtf(vi.val.w) + epsilon);

    vi.store(v + input.ev_start_indices + input.ev_id, 4);
    return gi.val;
  }

  DEVICE_INLINE float update(const OptimizierInput<wgrad_t> &input) {
    float vi =
        HugeCTR::TypeConvertFunc<float, acc_t>::convert(v[input.ev_start_indices + input.ev_id]);
    float gi = HugeCTR::TypeConvertFunc<float, wgrad_t>::convert(input.wgrad[input.ev_id]);
    gi = gi / input.scaler;
    vi = vi + gi * gi;

    gi = -input.lr * gi / (sqrtf(vi) + epsilon);
    v[input.ev_start_indices + input.ev_id] = HugeCTR::TypeConvertFunc<acc_t, float>::convert(vi);
    return gi;
  }
};

template <typename key_t, typename index_t, typename wgrad_t, typename OptimizerFunc,
          typename KeyToIndicesFunc>
__global__ void update4_kernel(const key_t *keys, const size_t *num_keys_ptr, const int *table_ids,
                               const wgrad_t *grad_ev, const uint32_t *ev_start_indices,
                               KeyToIndicesFunc key_to_indices_func, float *emb_table,
                               OptimizerFunc optimizer, float lr, float scaler) {
  size_t num_steps = (*num_keys_ptr - 1) / (blockDim.x * gridDim.x) + 1;
  for (size_t step = 0; step < num_steps; step++) {
    size_t tid = step * blockDim.x * gridDim.x + (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t emb_table_ev_start_indices_frag;
    int ev_size_frag = std::numeric_limits<int>::max();
    uint32_t grad_ev_offset_frag;
    if (tid < *num_keys_ptr) {
      key_t key = keys[tid];
      int table_id = table_ids[tid];
      key_to_indices_func(key, table_id, &emb_table_ev_start_indices_frag, &ev_size_frag);
      grad_ev_offset_frag = ev_start_indices[tid];
    }

    for (int lane_id = 0; lane_id < warpSize; lane_id++) {
      int ev_size = __shfl_sync(0xffffffff, ev_size_frag, lane_id);
      if (ev_size == std::numeric_limits<int>::max()) {
        break;
      }
      const wgrad_t *grad_ev_for_update =
          grad_ev + __shfl_sync(0xffffffff, grad_ev_offset_frag, lane_id);
      uint64_t ev_start_indices_v =
          __shfl_sync(0xffffffff, emb_table_ev_start_indices_frag, lane_id);
      float *ev = emb_table + ev_start_indices_v;

      for (int i = threadIdx.x % warpSize; i < ev_size / num_load_floats; i += warpSize) {
        OptimizierInput<wgrad_t> input{grad_ev_for_update, ev_start_indices_v, i * num_load_floats,
                                       lr, scaler};
        float4 gi = optimizer.update4(input);
        Vec4T<float> ev_plus_gi;
        ev_plus_gi.load(ev + i * num_load_floats, num_load_floats);

        ev_plus_gi.val.x += gi.x;
        ev_plus_gi.val.y += gi.y;
        ev_plus_gi.val.z += gi.z;
        ev_plus_gi.val.w += gi.w;

        ev_plus_gi.store(ev + i * 4, num_load_floats);
      }
    }
  }
}

template <typename key_t, typename index_t, typename emb_t, typename OptimizerFunc,
          typename KeyToIndicesFunc>
__global__ void update_kernel(const key_t *keys, const uint64_t *num_keys_ptr, const int *table_ids,
                              const emb_t *grad_ev, const uint32_t *ev_start_indices,
                              KeyToIndicesFunc key_to_indices_func, float *emb_table,
                              OptimizerFunc optimizer, float lr, float scaler) {
  uint64_t num_steps = (*num_keys_ptr - 1) / (blockDim.x * gridDim.x) + 1;
  for (size_t step = 0; step < num_steps; step++) {
    uint64_t tid = step * blockDim.x * gridDim.x + (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t emb_table_ev_start_indices_frag;
    int ev_size_frag = std::numeric_limits<int>::max();
    uint32_t grad_ev_offset_frag;
    if (tid < *num_keys_ptr) {
      key_t key = keys[tid];
      int table_id = table_ids[tid];
      key_to_indices_func(key, table_id, &emb_table_ev_start_indices_frag, &ev_size_frag);
      grad_ev_offset_frag = ev_start_indices[tid];
    }

    for (int lane_id = 0; lane_id < warpSize; lane_id++) {
      int ev_size = __shfl_sync(0xffffffff, ev_size_frag, lane_id);
      if (ev_size == std::numeric_limits<int>::max()) {
        break;
      }
      const emb_t *grad_ev_for_update =
          grad_ev + __shfl_sync(0xffffffff, grad_ev_offset_frag, lane_id);
      uint64_t ev_start_indices_v =
          __shfl_sync(0xffffffff, emb_table_ev_start_indices_frag, lane_id);
      float *ev = emb_table + ev_start_indices_v;

      for (int i = threadIdx.x % warpSize; i < ev_size; i += warpSize) {
        OptimizierInput<emb_t> input{grad_ev_for_update, ev_start_indices_v, i, lr, scaler};
        float gi = optimizer.update(input);
        ev[i] += gi;
      }
    }
  }
}

}  // namespace

RaggedStaticEmbeddingTable::RaggedStaticEmbeddingTable(
    const HugeCTR::GPUResource &gpu_resource, std::shared_ptr<CoreResourceManager> core,
    const std::vector<EmbeddingTableParam> &table_params, const EmbeddingCollectionParam &ebc_param,
    size_t grouped_id, const HugeCTR::OptParams &opt_param)
    : core_(core), emb_table_size_(0), use_vectorized_kernel_{true}, opt_param_(opt_param) {
  CudaDeviceContext ctx(core_->get_device_id());
  int global_gpu_id = core_->get_global_gpu_id();
  int num_gpus = core_->get_global_gpu_count();
  HCTR_CHECK_HINT(num_gpus == static_cast<int>(ebc_param.shard_matrix.size()),
                  "num_gpus is not match with shard matrix");

  auto key_type = ebc_param.key_type;
  auto index_type = ebc_param.index_type;
  auto emb_type = ebc_param.emb_type;
  const auto &emb_param = ebc_param.grouped_emb_params[grouped_id];
  for (const auto &table_param : table_params) {
    use_vectorized_kernel_ &= (table_param.ev_size % num_load_floats == 0);
  }

  DISPATCH_INTEGRAL_FUNCTION_CORE23(key_type.type(), key_t, [&] {
    DISPATCH_UNSIGNED_INTEGRAL_FUNCTION_CORE23(index_type.type(), index_t, [&] {
      std::vector<key_t> h_key_list;
      std::vector<index_t> h_num_key_per_table_offset{0};
      h_emb_table_ev_offset_.push_back(0);

      if (emb_param.table_placement_strategy == TablePlacementStrategy::DataParallel) {
        for (int table_id : emb_param.table_ids) {
          uint64_t num_key = 0;
          h_table_ids_.push_back(table_id);
          h_table_max_vocabulary_size_.push_back(table_params[table_id].max_vocabulary_size);
          for (int64_t k = 0; k < table_params[table_id].max_vocabulary_size; ++k) {
            h_key_list.push_back(k);
            num_key += 1;
          }
          h_num_key_per_table_.push_back(num_key);
          h_num_key_per_table_offset.push_back(num_key);

          uint64_t segment_emb_table_size = num_key * table_params[table_id].ev_size;
          h_size_per_table_.push_back(segment_emb_table_size);
          h_emb_table_ev_offset_.push_back(segment_emb_table_size);
          h_local_ev_sizes_.push_back(table_params[table_id].ev_size);
          emb_table_size_ += segment_emb_table_size;
        }
      } else if (emb_param.table_placement_strategy == TablePlacementStrategy::ModelParallel) {
        for (int table_id : emb_param.table_ids) {
          std::vector<int> shard_gpu_list;
          for (int gpu_id = 0; gpu_id < num_gpus; ++gpu_id) {
            HCTR_CHECK_HINT(table_id < static_cast<int>(ebc_param.shard_matrix[gpu_id].size()),
                            "table_id is out of range");
            if (ebc_param.shard_matrix[gpu_id][table_id] == 1) {
              shard_gpu_list.push_back(gpu_id);
            }
          }
          int num_shards = static_cast<int>(shard_gpu_list.size());
          auto find_shard_id_iter =
              std::find(shard_gpu_list.begin(), shard_gpu_list.end(), global_gpu_id);
          if (find_shard_id_iter == shard_gpu_list.end()) {
            continue;
          }
          uint64_t num_key = 0;
          h_table_ids_.push_back(table_id);
          h_table_max_vocabulary_size_.push_back(table_params[table_id].max_vocabulary_size);
          int shard_id =
              static_cast<int>(std::distance(shard_gpu_list.begin(), find_shard_id_iter));
          for (int64_t k = 0; k < table_params[table_id].max_vocabulary_size; ++k) {
            if (k % num_shards == shard_id) {
              h_key_list.push_back(k);
              num_key += 1;
            }
          }

          h_num_key_per_table_.push_back(num_key);
          h_num_key_per_table_offset.push_back(num_key);
          uint64_t segment_emb_table_size = num_key * table_params[table_id].ev_size;
          h_size_per_table_.push_back(segment_emb_table_size);
          h_emb_table_ev_offset_.push_back(segment_emb_table_size);
          h_local_ev_sizes_.push_back(table_params[table_id].ev_size);
          emb_table_size_ += segment_emb_table_size;
        }
      }

      std::partial_sum(h_num_key_per_table_offset.begin(), h_num_key_per_table_offset.end(),
                       h_num_key_per_table_offset.begin());
      std::partial_sum(h_emb_table_ev_offset_.begin(), h_emb_table_ev_offset_.end(),
                       h_emb_table_ev_offset_.begin());
      for (auto tmp_offset : h_num_key_per_table_offset) {
        h_num_key_per_table_offset_.push_back(static_cast<size_t>(tmp_offset));
      }

      core23::Device device(core23::DeviceType::GPU, core->get_device_id());
      core23::TensorParams params = core23::TensorParams().device(device);

      table_ids_ = core23::Tensor(params.shape({static_cast<int64_t>(h_table_ids_.size())})
                                      .data_type(core23::ScalarType::Int32));
      keys_ = core23::Tensor(
          params.shape({static_cast<int64_t>(h_key_list.size())}).data_type(key_type));
      num_key_per_table_offset_ =
          core23::Tensor(params.shape({static_cast<int64_t>(h_num_key_per_table_offset.size())})
                             .data_type(index_type));
      emb_table_ = core23::Tensor(params.shape({static_cast<int64_t>(emb_table_size_)})
                                      .data_type(core23::ScalarType::Float));
      emb_table_ev_offset_ =
          core23::Tensor(params.shape({static_cast<int64_t>(h_emb_table_ev_offset_.size())})
                             .data_type(core23::ScalarType::UInt64));
      local_ev_size_list_ =
          core23::Tensor(params.shape({static_cast<int64_t>(h_local_ev_sizes_.size())})
                             .data_type(core23::ScalarType::Int32));

      core23::copy_sync(table_ids_, h_table_ids_);
      core23::copy_sync(keys_, h_key_list);
      core23::copy_sync(num_key_per_table_offset_, h_num_key_per_table_offset);
      core23::copy_sync(emb_table_ev_offset_, h_emb_table_ev_offset_);
      core23::copy_sync(local_ev_size_list_, h_local_ev_sizes_);
    });
  });

  if (opt_param.optimizer == HugeCTR::Optimizer_t::AdaGrad) {
    DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(emb_type.type(), emb_t, [&] {
      core23::Device device(core23::DeviceType::GPU, core->get_device_id());
      core23::TensorParams params = core23::TensorParams().device(device);
      auto accum_tensor = core23::Tensor(params.shape({static_cast<int64_t>(emb_table_size_)})
                                             .data_type(core23::ScalarType::Float));

      HCTR_LIB_THROW(hipMemset(accum_tensor.data(), 0, accum_tensor.num_bytes()));
      opt_buffer_ = AdaGradOptBuffer{accum_tensor};
    });
  }

  for (size_t i = 0; i < h_table_ids_.size(); i++) {
    int table_id = h_table_ids_[i];
    std::function<void(const hiprandGenerator_t &)> init_table_functor;

    if (table_params[table_id].init_param.initializer_type == HugeCTR::Initializer_t::Default) {
      init_table_functor = [&](const hiprandGenerator_t &generator) {
        float up_bound = sqrt(1.f / h_table_max_vocabulary_size_[i]);
        size_t offset = h_emb_table_ev_offset_[i];
        size_t num_elements = h_emb_table_ev_offset_[i + 1] - h_emb_table_ev_offset_[i];

        HugeCTR::UniformGenerator::fill(emb_table_.data<float>() + offset, num_elements, -up_bound,
                                        up_bound, gpu_resource.get_sm_count(), generator,
                                        gpu_resource.get_stream());
      };
    } else if (table_params[table_id].init_param.initializer_type ==
               HugeCTR::Initializer_t::Uniform) {
      init_table_functor = [&](const hiprandGenerator_t &generator) {
        float up_bound = table_params[table_id].init_param.uniform_params.up_bound;
        size_t offset = h_emb_table_ev_offset_[i];
        size_t num_elements = h_emb_table_ev_offset_[i + 1] - h_emb_table_ev_offset_[i];

        HugeCTR::UniformGenerator::fill(emb_table_.data<float>() + offset, num_elements, -up_bound,
                                        up_bound, gpu_resource.get_sm_count(), generator,
                                        gpu_resource.get_stream());
      };
    } else if (table_params[table_id].init_param.initializer_type ==
               HugeCTR::Initializer_t::Sinusoidal) {
      init_table_functor = [&](const hiprandGenerator_t &) {
        const SinusoidalParams &sinus_params = table_params[table_id].init_param.sinusoidal_params;
        int max_sequence_len = sinus_params.max_sequence_len;
        int ev_size = sinus_params.ev_size;
        size_t offset = h_emb_table_ev_offset_[i];
        size_t num_elements = h_emb_table_ev_offset_[i + 1] - h_emb_table_ev_offset_[i];

        HCTR_CHECK_HINT(max_sequence_len * ev_size == static_cast<int>(num_elements),
                        "max_sequent_len * ev_size %d should equal to num_elements %d",
                        max_sequence_len * ev_size, static_cast<int>(num_elements));
        HugeCTR::SinusoidalGenerator::fill(emb_table_.data<float>() + offset, num_elements, ev_size,
                                           max_sequence_len, gpu_resource.get_sm_count(),
                                           gpu_resource.get_stream());
      };
    } else {
      HCTR_OWN_THROW(HugeCTR::Error_t::IllegalCall, "initializer not implemented");
    }

    // data parallel table should use same hiprand seed across all gpus
    if (emb_param.table_placement_strategy == TablePlacementStrategy::DataParallel) {
      init_table_functor(gpu_resource.get_replica_uniform_curand_generator());
    } else {
      init_table_functor(gpu_resource.get_replica_variant_curand_generator());
    }
  }
}

void RaggedStaticEmbeddingTable::lookup(const core23::Tensor &keys, size_t num_keys,
                                        const core23::Tensor &id_space_offset,
                                        size_t num_id_space_offset,
                                        const core23::Tensor &id_space_list,
                                        core23::Tensor &emb_vec) {
  CudaDeviceContext ctx(core_->get_device_id());

  DISPATCH_INTEGRAL_FUNCTION_CORE23(keys.data_type().type(), key_t, [&] {
    DISPATCH_UNSIGNED_INTEGRAL_FUNCTION_CORE23(
        num_key_per_table_offset_.data_type().type(), index_t, [&] {
          hipStream_t stream = core_->get_local_gpu()->get_stream();

          if (num_keys > 0) {  // batch size is small there can be situation that we do not need
                               // have key for lookup
            constexpr int block_size = 256;
            int grid_size = (num_keys - 1) / block_size + 1;
            ragged_static_embedding_table_lookup_kernel<<<grid_size, block_size, 0, stream>>>(
                keys.data<key_t>(), num_keys, id_space_offset.data<uint32_t>(), num_id_space_offset,
                id_space_list.data<int>(), table_ids_.data<int>(), table_ids_.num_elements(),
                num_key_per_table_offset_.data<index_t>(), emb_table_.data<float>(),
                emb_table_ev_offset_.data<uint64_t>(), local_ev_size_list_.data<int>(),
                static_cast<float **>(emb_vec.data()));
          }

          HCTR_LIB_THROW(hipPeekAtLastError());
        });
  });
}

void RaggedStaticEmbeddingTable::update(const core23::Tensor &unique_keys,
                                        const core23::Tensor &num_unique_keys,
                                        const core23::Tensor &table_ids,
                                        const core23::Tensor &ev_start_indices,
                                        const core23::Tensor &wgrad) {
  CudaDeviceContext context(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();

  HCTR_CHECK_HINT(opt_param_.optimizer != HugeCTR::Optimizer_t::NOT_INITIALIZED,
                  "optimizer not initialized");
  HCTR_CHECK(num_unique_keys.data_type() == core23::ScalarType::UInt64);
  HCTR_CHECK(table_ids.data_type() == core23::ScalarType::Int32);
  HCTR_CHECK(ev_start_indices.data_type() == core23::ScalarType::UInt32);
  HCTR_CHECK(wgrad.data_type() == core23::ScalarType::Float);

  if (opt_param_.optimizer == HugeCTR::Optimizer_t::SGD) {
    DISPATCH_INTEGRAL_FUNCTION_CORE23(unique_keys.data_type().type(), key_t, [&] {
      DISPATCH_UNSIGNED_INTEGRAL_FUNCTION_CORE23(
          num_key_per_table_offset_.data_type().type(), index_t, [&] {
            DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.data_type().type(), wgrad_t, [&] {
              RaggedKeyToIndicesFunc<key_t, index_t> key_to_indices_func{
                  table_ids_.data<int>(),
                  local_ev_size_list_.data<int>(),
                  table_ids_.num_elements(),
                  num_key_per_table_offset_.data<index_t>(),
                  emb_table_ev_offset_.data<uint64_t>(),
              };
              SGDOptimizer<wgrad_t> optimizer;

              constexpr int block_size = 256;
              constexpr int grid_size = 144 * 8;
              auto kernel = use_vectorized_kernel_
                                ? update4_kernel<key_t, index_t, wgrad_t, decltype(optimizer),
                                                 decltype(key_to_indices_func)>
                                : update_kernel<key_t, index_t, wgrad_t, decltype(optimizer),
                                                decltype(key_to_indices_func)>;
              kernel<<<grid_size, block_size, 0, stream>>>(
                  unique_keys.data<key_t>(), num_unique_keys.data<size_t>(), table_ids.data<int>(),
                  wgrad.data<wgrad_t>(), ev_start_indices.data<uint32_t>(), key_to_indices_func,
                  emb_table_.data<float>(), optimizer, opt_param_.lr, opt_param_.scaler);
            });
          });
    });
  } else if (opt_param_.optimizer == HugeCTR::Optimizer_t::AdaGrad) {
    DISPATCH_INTEGRAL_FUNCTION_CORE23(unique_keys.data_type().type(), key_t, [&] {
      DISPATCH_UNSIGNED_INTEGRAL_FUNCTION_CORE23(
          num_key_per_table_offset_.data_type().type(), index_t, [&] {
            DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(wgrad.data_type().type(), wgrad_t, [&] {
              auto adagrad_opt_buffer = std::get_if<AdaGradOptBuffer>(&opt_buffer_);
              HCTR_CHECK_HINT(adagrad_opt_buffer != nullptr, "Adagrad Opt Buffer not initialized.");
              DISPATCH_FLOAT_AND_HALF_FUNCTION_CORE23(
                  adagrad_opt_buffer->opt_accum_tensor.data_type().type(), acc_t, [&] {
                    RaggedKeyToIndicesFunc<key_t, index_t> key_to_indices_func{
                        table_ids_.data<int>(),
                        local_ev_size_list_.data<int>(),
                        table_ids_.num_elements(),
                        num_key_per_table_offset_.data<index_t>(),
                        emb_table_ev_offset_.data<uint64_t>(),
                    };
                    AdaGradOptimizer<wgrad_t, acc_t> optimizer{
                        adagrad_opt_buffer->opt_accum_tensor.data<acc_t>(),
                        opt_param_.hyperparams.adagrad.epsilon};

                    constexpr int block_size = 256;
                    constexpr int grid_size = 8 * 144;
                    auto kernel = use_vectorized_kernel_
                                      ? update4_kernel<key_t, index_t, wgrad_t, decltype(optimizer),
                                                       decltype(key_to_indices_func)>
                                      : update_kernel<key_t, index_t, wgrad_t, decltype(optimizer),
                                                      decltype(key_to_indices_func)>;
                    kernel<<<grid_size, block_size, 0, stream>>>(
                        unique_keys.data<key_t>(), num_unique_keys.data<size_t>(),
                        table_ids.data<int>(), wgrad.data<wgrad_t>(),
                        ev_start_indices.data<uint32_t>(), key_to_indices_func,
                        emb_table_.data<float>(), optimizer, opt_param_.lr, opt_param_.scaler);
                  });
            });
          });
    });
  } else {
    HCTR_OWN_THROW(HugeCTR::Error_t::IllegalCall, "optimizer not implemented");
  }
}
}  // namespace embedding
