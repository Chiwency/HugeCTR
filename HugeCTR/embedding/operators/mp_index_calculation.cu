/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hipcub/hipcub.hpp>

#include "HugeCTR/include/utils.cuh"
#include "HugeCTR/include/utils.hpp"
#include "generic_lookup.cuh"
#include "mp_index_calculation.hpp"

namespace embedding {
void MPLocalReduceIndexCalculation::init(
    std::shared_ptr<CoreResourceManager> core,
    const LocalReduceIndexCalculation& local_reduce_index_calculation,
    const SegmentedSortDevice& segmented_sort_device, const CalDstIds& cal_dst_ids,
    const SegmentdUnique& segmented_unique, const CalDstOffsetMP& cal_dst_offset_mp) {
  core_ = core;
  local_reduce_index_calculation_ = local_reduce_index_calculation;
  segmented_sort_device_ = segmented_sort_device;
  cal_dst_ids_ = cal_dst_ids;
  segmented_unique_ = segmented_unique;
  cal_dst_offset_mp_ = cal_dst_offset_mp;
}

void MPLocalReduceIndexCalculation::init(
    std::shared_ptr<CoreResourceManager> core,
    const LocalReduceIndexCalculation& local_reduce_index_calculation,
    const IndicesSort& indices_sort, const CalDstIds& cal_dst_ids,
    const SegmentdUnique& segmented_unique, const CalDstOffsetMP& cal_dst_offset_mp) {
  core_ = core;
  local_reduce_index_calculation_ = local_reduce_index_calculation;
  indices_sort_ = indices_sort;
  cal_dst_ids_ = cal_dst_ids;
  segmented_unique_ = segmented_unique;
  cal_dst_offset_mp_ = cal_dst_offset_mp;
}

void MPLocalReduceIndexCalculation::cal_for_sparse_input(const EmbeddingInput& embedding_input,
                                                         ReductionIndices& reduction_indices,
                                                         Wgrad& wgrad, int batch_size,
                                                         bool need_cal_unique_range) {
  auto cal_ev_start_indices_in_local_wgrad = [&](const WgradEvStartIndicesCalculationInput& input,
                                                 WgradEvStartIndicesCalculationOutput& output,
                                                 hipStream_t stream) {
    cal_dst_offset_mp_(input.table_ids, input.table_id_to_ev_size, input.num_unique_keys,
                       output.ev_start_indices, stream);
  };
  if (!indices_sort_.table_id_to_global_start_indices.empty()) {
    local_reduce_index_calculation_.cal_for_sparse_input(embedding_input, indices_sort_,
                                                         segmented_unique_, cal_dst_ids_,
                                                         reduction_indices, wgrad, batch_size);
    if (need_cal_unique_range) local_reduce_index_calculation_.cal_unique_key_table_range(wgrad);
    local_reduce_index_calculation_.cal_dst_ev_start(wgrad, cal_ev_start_indices_in_local_wgrad);
  } else {
    local_reduce_index_calculation_.cal_for_sparse_input(embedding_input, segmented_sort_device_,
                                                         segmented_unique_, cal_dst_ids_,
                                                         reduction_indices, wgrad, batch_size);
    if (need_cal_unique_range) local_reduce_index_calculation_.cal_unique_key_table_range(wgrad);
    local_reduce_index_calculation_.cal_dst_ev_start(wgrad, cal_ev_start_indices_in_local_wgrad);
  }
}

}  // namespace embedding
