/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <embedding/common.hpp>
#include <embedding/operators/generic_lookup.cuh>
#include <embedding/operators/network_backward.hpp>
#include <embedding/operators/network_forward.hpp>
#include <utils.hpp>

namespace embedding {

using namespace core;
namespace {

void network_backward_from_feature_major_top_grad(const Tensor& bucket_range,
                                                  const EmbeddingOutput& top_grad,
                                                  const NetworkIndices& network_indices,
                                                  NetworkBuffer& network_buffer, int batch_size,
                                                  int gpu_id, int num_gpus, hipStream_t stream) {
  auto& top_grad_attr = top_grad.attr;
  auto& network_attr = network_buffer.attr;
  int batch_size_per_gpu = batch_size / num_gpus;
  int max_ev_size = top_grad_attr.max_ev_size;

  DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION(top_grad.data.dtype().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION(network_attr.type.type(), dst_emb_t, [&] {
        const offset_t* bucket_range_ptr = bucket_range.get<offset_t>();
        const int* network_ids_ptr = network_indices.network_ids.get<int>();
        const int* network_gpu_ids_ptr = network_indices.network_gpu_ids.get<int>();
        const int* network_offsets_ptr = network_indices.network_offsets.get<int>();
        const int* network_dst_lookup_ids_ptr = network_indices.network_dst_lookup_ids.get<int>();
        int** network_ev_sizes_ptr = network_attr.id_to_ev_size.get<int>();
        int** network_ev_offsets_ptr = network_attr.id_to_ev_start_indices.get<int>();
        const int* d_ev_size_offset_ptr = top_grad_attr.id_to_ev_start_indices.get<int>();
        const emb_t* top_grad_ptr = top_grad.data.get<emb_t>();
        dst_emb_t** network_comm_buffer_ptr = network_buffer.data.get<dst_emb_t>();
        const char* combiner_ptr = top_grad_attr.id_to_combiner.get<char>();
        int num_network_dst_lookup_ids = network_indices.network_dst_lookup_ids.get_num_elements();

        auto one_to_multi_desc = make_MultiToOne<emb_t, dst_emb_t>(
            num_network_dst_lookup_ids * batch_size_per_gpu,
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = i % num_network_dst_lookup_ids;
              return bid * network_offsets_ptr[num_network_dst_lookup_ids] +
                     network_offsets_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
                int start = batch_size * lookup_id + gpu_id * batch_size_per_gpu + bid;
                return static_cast<int>(bucket_range_ptr[start + 1] - bucket_range_ptr[start]);
              } else {
                return 1;
              }
            },
            [=] __device__(int i) {
              int dst_lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              return d_ev_size_offset_ptr[dst_lookup_id + 1] - d_ev_size_offset_ptr[dst_lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              int ev_offset = d_ev_size_offset_ptr[lookup_id] * batch_size_per_gpu;
              int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
              return top_grad_ptr + ev_offset + bid * ev_size;
            },
            [=] __device__(int i) {
              int bid = i / network_offsets_ptr[num_network_dst_lookup_ids];
              int id = i % network_offsets_ptr[num_network_dst_lookup_ids];

              int network_gpu_id = network_gpu_ids_ptr[id];
              int network_id = network_ids_ptr[id];
              int ev_offset =
                  network_ev_offsets_ptr[network_gpu_id][network_id] * batch_size_per_gpu;
              int ev_size = network_ev_sizes_ptr[network_gpu_id][network_id];

              return network_comm_buffer_ptr[network_gpu_id] + ev_offset + bid * ev_size;
            });
        copy_one_to_multi(one_to_multi_desc, max_ev_size, stream);
      });
    });
  });
}

void network_backward_from_batch_major_top_grad(const Tensor& bucket_range,
                                                const EmbeddingOutput& top_grad,
                                                const NetworkIndices& network_indices,
                                                NetworkBuffer& network_buffer, int batch_size,
                                                int gpu_id, int num_gpus, hipStream_t stream) {
  auto& top_grad_attr = top_grad.attr;
  auto& network_attr = network_buffer.attr;
  int batch_size_per_gpu = batch_size / num_gpus;
  int max_ev_size = top_grad_attr.max_ev_size;
  int num_lookup = top_grad_attr.id_to_ev_size.get_num_elements();

  DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION(top_grad.data.dtype().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION(network_attr.type.type(), dst_emb_t, [&] {
        const offset_t* bucket_range_ptr = bucket_range.get<offset_t>();
        const int* network_ids_ptr = network_indices.network_ids.get<int>();
        const int* network_gpu_ids_ptr = network_indices.network_gpu_ids.get<int>();
        const int* network_offsets_ptr = network_indices.network_offsets.get<int>();
        const int* network_dst_lookup_ids_ptr = network_indices.network_dst_lookup_ids.get<int>();
        int** network_ev_sizes_ptr = network_attr.id_to_ev_size.get<int>();
        int** network_ev_offsets_ptr = network_attr.id_to_ev_start_indices.get<int>();
        const int* d_ev_size_offset_ptr = top_grad_attr.id_to_ev_start_indices.get<int>();
        const emb_t* top_grad_ptr = top_grad.data.get<emb_t>();
        dst_emb_t** network_comm_buffer_ptr = network_buffer.data.get<dst_emb_t>();
        const char* combiner_ptr = top_grad_attr.id_to_combiner.get<char>();
        int num_network_dst_lookup_ids = network_indices.network_dst_lookup_ids.get_num_elements();

        auto one_to_multi_desc = make_MultiToOne<emb_t, dst_emb_t>(
            num_network_dst_lookup_ids * batch_size_per_gpu,
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = i % num_network_dst_lookup_ids;
              return bid * network_offsets_ptr[num_network_dst_lookup_ids] +
                     network_offsets_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
                int start = batch_size * lookup_id + gpu_id * batch_size_per_gpu + bid;
                return static_cast<int>(bucket_range_ptr[start + 1] - bucket_range_ptr[start]);
              } else {
                return 1;
              }
            },
            [=] __device__(int i) {
              int dst_lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              return d_ev_size_offset_ptr[dst_lookup_id + 1] - d_ev_size_offset_ptr[dst_lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              int ev_offset = d_ev_size_offset_ptr[num_lookup] * bid;
              int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
              return top_grad_ptr + ev_offset + d_ev_size_offset_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / network_offsets_ptr[num_network_dst_lookup_ids];
              int id = i % network_offsets_ptr[num_network_dst_lookup_ids];

              int network_gpu_id = network_gpu_ids_ptr[id];
              int network_id = network_ids_ptr[id];
              int ev_offset =
                  network_ev_offsets_ptr[network_gpu_id][network_id] * batch_size_per_gpu;
              int ev_size = network_ev_sizes_ptr[network_gpu_id][network_id];

              return network_comm_buffer_ptr[network_gpu_id] + ev_offset + bid * ev_size;
            });
        copy_one_to_multi(one_to_multi_desc, max_ev_size, stream);
      });
    });
  });
}

}  // namespace
void NetworkBackward::compute(const Tensor& bucket_range, const EmbeddingOutput& top_grad,
                              const NetworkIndices& network_indices, NetworkBuffer& network_buffer,
                              int batch_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  auto stream = core_->get_local_gpu()->get_stream();
  int gpu_id = core_->get_global_gpu_id();
  int num_gpus = core_->get_global_gpu_count();

  if (top_grad.attr.layout == EmbeddingLayout::FeatureMajor) {
    network_backward_from_feature_major_top_grad(bucket_range, top_grad, network_indices,
                                                 network_buffer, batch_size, gpu_id, num_gpus,
                                                 stream);
  } else {
    HCTR_ASSERT(top_grad.attr.layout == EmbeddingLayout::BatchMajor);
    network_backward_from_batch_major_top_grad(bucket_range, top_grad, network_indices,
                                               network_buffer, batch_size, gpu_id, num_gpus,
                                               stream);
  }
}

void NetworkBackward::compute(const TensorList& row_lengths, const Tensor& d_combiner_list,
                              const TensorList& top_grad, const Tensor& network_ids,
                              const Tensor& network_gpu_ids, const Tensor& network_offsets,
                              const Tensor& network_dst_lookup_ids,
                              const TensorList& network_ev_sizes,
                              const TensorList& network_ev_offsets, TensorList& network_comm_buffer,
                              const Tensor& d_ev_size_offset, int batch_size, int max_ev_size) {
  HugeCTR::CudaDeviceContext ctx(core_->get_device_id());
  int batch_size_per_gpu = batch_size / num_gpus_;
  auto stream = core_->get_local_gpu()->get_stream();

  DISPATCH_INTEGRAL_FUNCTION(row_lengths.dtype().type(), offset_t, [&] {
    DISPATCH_FLOAT_AND_HALF_FUNCTION(top_grad.dtype().type(), emb_t, [&] {
      DISPATCH_FLOAT_AND_HALF_FUNCTION(network_comm_buffer.dtype().type(), dst_emb_t, [&] {
        const offset_t** row_lengths_ptr = row_lengths.get<offset_t>();
        const int* network_ids_ptr = network_ids.get<int>();
        const int* network_gpu_ids_ptr = network_gpu_ids.get<int>();
        const int* network_offsets_ptr = network_offsets.get<int>();
        const int* network_dst_lookup_ids_ptr = network_dst_lookup_ids.get<int>();
        const int** network_ev_sizes_ptr = network_ev_sizes.get<int>();
        const int** network_ev_offsets_ptr = network_ev_offsets.get<int>();
        const int* d_ev_size_offset_ptr = d_ev_size_offset.get<int>();
        const emb_t** top_grad_ptr = top_grad.get<emb_t>();
        dst_emb_t** network_comm_buffer_ptr = network_comm_buffer.get<dst_emb_t>();
        const char* combiner_ptr = d_combiner_list.get<char>();
        int num_network_dst_lookup_ids = network_dst_lookup_ids.get_num_elements();
        int gpu_id = core_->get_global_gpu_id();

        auto one_to_multi_desc = make_MultiToOne<emb_t, dst_emb_t>(
            num_network_dst_lookup_ids * batch_size_per_gpu,
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = i % num_network_dst_lookup_ids;
              return bid * network_offsets_ptr[num_network_dst_lookup_ids] +
                     network_offsets_ptr[lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              if (combiner_ptr[lookup_id] == static_cast<char>(Combiner::Average)) {
                return static_cast<int>(row_lengths_ptr[lookup_id][bid]);
              } else {
                return 1;
              }
            },
            [=] __device__(int i) {
              int dst_lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];
              return d_ev_size_offset_ptr[dst_lookup_id + 1] - d_ev_size_offset_ptr[dst_lookup_id];
            },
            [=] __device__(int i) {
              int bid = i / num_network_dst_lookup_ids;
              int lookup_id = network_dst_lookup_ids_ptr[i % num_network_dst_lookup_ids];

              int ev_size = d_ev_size_offset_ptr[lookup_id + 1] - d_ev_size_offset_ptr[lookup_id];
              return top_grad_ptr[lookup_id] + bid * ev_size;
            },
            [=] __device__(int i) {
              int bid = i / network_offsets_ptr[num_network_dst_lookup_ids];
              int id = i % network_offsets_ptr[num_network_dst_lookup_ids];

              int network_gpu_id = network_gpu_ids_ptr[id];
              int network_id = network_ids_ptr[id];
              int ev_offset =
                  network_ev_offsets_ptr[network_gpu_id][network_id] * batch_size_per_gpu;
              int ev_size = network_ev_sizes_ptr[network_gpu_id][network_id];

              return network_comm_buffer_ptr[network_gpu_id] + ev_offset + bid * ev_size;
            });
        copy_one_to_multi(one_to_multi_desc, max_ev_size, stream);
      });
    });
  });
}
}  // namespace embedding
