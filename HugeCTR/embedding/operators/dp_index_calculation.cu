#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include <embedding/operators/dp_index_calculation.hpp>
#include <embedding/operators/generic_lookup.cuh>
#include <utils.cuh>
#include <utils.hpp>

namespace embedding {
namespace {

template <typename key_t>
__global__ void cal_ev_start_indices_in_allreduce_wgrad_using_indices_kernel(
    const key_t* unique_indices, int num_elements, const uint32_t* table_range,
    const int* unique_table_ids, const int* table_id_to_allreduce_buffer_start_indices,
    const uint32_t* ev_start_indices_in_allreduce_buffer, const size_t* num_unique_key,
    int vocabulary_size, uint32_t* ev_start_indices_for_local_reduce) {
  uint32_t num_keys = static_cast<uint32_t>(*num_unique_key);
  CUDA_1D_KERNEL_LOOP_T(uint32_t, i, num_elements) {
    if (i >= num_keys) {
      ev_start_indices_for_local_reduce[i] = 0;
      continue;
    }
    uint32_t idx = i;

    int table_id = unique_table_ids[idx];
    int idx_in_allreduce_buffer = static_cast<int>(unique_indices[idx]) +
                                  table_id_to_allreduce_buffer_start_indices[table_id];

    ev_start_indices_for_local_reduce[i] =
        ev_start_indices_in_allreduce_buffer[idx_in_allreduce_buffer];
  }
}
}  // namespace

void DenseAllreduceIndexCalculation::cal_for_sparse_indices(
    const EmbeddingInput& embedding_input, const Tensor& table_id_to_allreduce_buffer_start_indices,
    const Tensor& ev_start_indices_in_allreduce_buffer, ReductionIndices& reduction_indices,
    Wgrad& wgrad, int batch_size) {
  int gpu_id = core_->get_global_gpu_id();
  int num_gpus = core_->get_global_gpu_count();

  auto cal_ev_start_indices_in_allreduce_wgrad =
      [&](const WgradEvStartIndicesCalculationInput& input,
          WgradEvStartIndicesCalculationOutput& output, hipStream_t stream) {
        auto key_type = input.unique_keys.dtype();

        DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
          cal_ev_start_indices_in_allreduce_wgrad_using_indices_kernel<<<144 * 8, 256, 0, stream>>>(
              input.unique_keys.get<key_t>(), input.unique_keys.get_num_elements(),
              input.table_range.get<uint32_t>(), input.table_ids.get<int>(),
              table_id_to_allreduce_buffer_start_indices.get<int>(),
              ev_start_indices_in_allreduce_buffer.get<uint32_t>(),
              input.num_unique_keys.get<size_t>(),
              ev_start_indices_in_allreduce_buffer.get_num_elements() - 1,
              output.ev_start_indices.get<uint32_t>());
        });
      };

  local_reduce_index_calculation_.cal_for_sparse_input(embedding_input, indices_sort_,
                                                       segmented_unique_, cal_dst_ids_,
                                                       reduction_indices, wgrad, batch_size);
  local_reduce_index_calculation_.cal_dst_ev_start(wgrad, cal_ev_start_indices_in_allreduce_wgrad);
}

SparseAllreduceCalEVStartIndicesStorage::SparseAllreduceCalEVStartIndicesStorage(
    std::shared_ptr<CoreResourceManager> core, int num_table, int local_hotness_sum, int batch_size,
    DataType key_type) {
  auto buffer_ptr = core::GetBuffer(core);
  int num_gpus = core->get_global_gpu_count();

  // BroadcastResult
  broadcast_result_.allgather_table_range_ = buffer_ptr->reserve(
      {num_gpus * (num_table + 1)}, core::DeviceType::GPU, core::TensorScalarType::UInt32);
  broadcast_result_.h_table_range_ =
      buffer_ptr->reserve({num_table + 1}, core::DeviceType::CPU, core::TensorScalarType::UInt32);
  broadcast_result_.reordered_allgather_table_range_ = buffer_ptr->reserve(
      {num_gpus * num_table + 1}, core::DeviceType::GPU, core::TensorScalarType::UInt32);
  broadcast_result_.h_reordered_allgather_table_range_ = buffer_ptr->reserve(
      {num_gpus * num_table + 1}, core::DeviceType::CPU, core::TensorScalarType::UInt32);
  broadcast_result_.allgather_unique_keys_ = buffer_ptr->reserve(
      {local_hotness_sum * batch_size * num_gpus}, core::DeviceType::GPU, key_type);

  // HashTable
  DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
    hash_table_.hash_table_ =
        buffer_ptr->reserve({local_hotness_sum * batch_size * num_gpus * sizeof(TableEntry<key_t>)},
                            core::DeviceType::GPU, core::TensorScalarType::Void);
  });
  {
    size_t temp_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                  num_table + 1);
    hash_table_.d_temp_scan_table_range_storage_ =
        buffer_ptr->reserve({temp_bytes}, core::DeviceType::GPU, TensorScalarType::Void);
  }

  // Tempstorage
  {
    size_t temp_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                  local_hotness_sum * batch_size * num_gpus + 1);
    temp_storage_.d_temp_scan_ev_start_indices_storage_ =
        buffer_ptr->reserve({temp_bytes}, core::DeviceType::GPU, TensorScalarType::Void);
  }
  temp_storage_.mask_unique_keys_in_allgather_unique_keys_ = buffer_ptr->reserve(
      {local_hotness_sum * batch_size * num_gpus}, core::DeviceType::GPU, TensorScalarType::Int32);
  {
    size_t temp_bytes = 0;
    DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
      hipcub::DeviceSelect::Flagged(nullptr, temp_bytes, (key_t*)nullptr, (int*)nullptr,
                                 (key_t*)nullptr, (size_t*)nullptr,
                                 local_hotness_sum * batch_size * num_gpus + 1);
    });
    temp_storage_.d_temp_select_temp_storage_ =
        buffer_ptr->reserve({temp_bytes}, core::DeviceType::GPU, TensorScalarType::Void);
  }
  {
    size_t temp_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_bytes, (uint32_t*)nullptr, (uint32_t*)nullptr,
                                  local_hotness_sum * batch_size * num_gpus + 1);
    temp_storage_.d_temp_scan_unique_idx_temp_storage_ =
        buffer_ptr->reserve({temp_bytes}, core::DeviceType::GPU, TensorScalarType::Void);
  }
  temp_storage_.unique_idx_ = buffer_ptr->reserve({local_hotness_sum * batch_size * num_gpus},
                                                  core::DeviceType::GPU, TensorScalarType::Int32);
  buffer_ptr->allocate();
}

namespace {
// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp {
  // Running prefix
  int running_total;
  // Constructor
  explicit __device__ BlockPrefixCallbackOp(int running_total) : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide scan.
  __device__ int operator()(int block_aggregate) {
    int old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <int TPB>
__global__ void reorder_allgather_table_range_kernel(const uint32_t* allgather_table_range,
                                                     int num_table, int num_gpus,
                                                     uint32_t* reordered_table_range) {
  typedef hipcub::BlockScan<uint32_t, TPB> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  CUDA_1D_KERNEL_LOOP(i, num_table * num_gpus + 1) {
    uint32_t num_unique_keys = 0;
    if (i > 0) {
      int gpu_id = (i - 1) % num_gpus;
      int table_id = (i - 1) / num_gpus;

      num_unique_keys = allgather_table_range[gpu_id * (num_table + 1) + table_id + 1] -
                        allgather_table_range[gpu_id * (num_table + 1) + table_id];
    }

    uint32_t start;
    BlockScan(temp_storage).InclusiveSum(num_unique_keys, start, prefix_op);
    __syncthreads();

    reordered_table_range[i] = start;
  }
}

struct DirectHash {
  DEVICE_INLINE uint32_t operator()(int32_t v) { return static_cast<uint32_t>(v); }
  DEVICE_INLINE uint32_t operator()(uint32_t v) { return static_cast<uint32_t>(v); }
  DEVICE_INLINE uint32_t operator()(int64_t v) { return static_cast<uint32_t>(v); }
  DEVICE_INLINE uint32_t operator()(uint64_t v) { return static_cast<uint32_t>(v); }
};

// this kernel does:
template <typename key_t, typename HASH>
__global__ void hash_table_insert_key_and_index_kernel(const key_t* allgather_unique_keys,
                                                       const uint32_t* allgather_table_range,
                                                       int num_table, int num_gpus,
                                                       TableEntry<key_t>* table,
                                                       uint32_t* unique_keys_table_range) {
  for (int ith_gpu = blockIdx.y; ith_gpu < num_gpus; ith_gpu += gridDim.y) {
    for (int ith_table = blockIdx.x; ith_table < num_table; ith_table += gridDim.x) {
      uint32_t range_table_start = allgather_table_range[ith_table * num_gpus + ith_gpu];
      uint32_t range_table_end = allgather_table_range[ith_table * num_gpus + ith_gpu + 1];
      uint32_t num_keys_in_table = range_table_end - range_table_start;
      uint32_t table_capacity = allgather_table_range[(ith_table + 1) * num_gpus] -
                                allgather_table_range[ith_table * num_gpus];

      auto current_table = table + allgather_table_range[ith_table * num_gpus];
      for (uint32_t i = threadIdx.x; i < num_keys_in_table; i += blockDim.x) {
        uint32_t idx = range_table_start + i;
        uint32_t idx_plus_one = idx + 1;
        key_t key = allgather_unique_keys[idx];
        uint32_t key_hash = HASH()(key);
        uint32_t pos = key_hash % table_capacity;

        const key_t key_hi = (key | 0x1);
        const auto key_lo = static_cast<uint32_t>(key & 0x1);
        bool finish_insert = false;
        while (!finish_insert) {
          bool prob_next = false;
          key_t* key_ptr = &current_table[pos].key;
          volatile uint32_t* table_value_ptr = &current_table[pos].value;

          const key_t old_key = atomicCAS(key_ptr, 0, key_hi);
          if (old_key == 0) {
            *table_value_ptr = (idx_plus_one << 1U | key_lo);
            atomicAdd(unique_keys_table_range + 1 + ith_table, 1);
            finish_insert = true;
          } else if (old_key == key_hi) {
            const uint32_t value = *table_value_ptr;
            if (value == 0) {
              // do nothing.
            } else if ((value & 0x1) == key_lo) {
              if ((value >> 1U) > idx_plus_one) {
                // substitution with smaller idx
                *table_value_ptr = (idx_plus_one << 1U | key_lo);
              } else {
                // old idx is smaller. do nothing
              }
              finish_insert = true;
            } else {
              prob_next = true;
            }
          } else {
            prob_next = true;
          }
          if (prob_next) {
            pos += 1;
            if (pos >= table_capacity) {
              pos -= table_capacity;
            }
          }
        }
      }
    }
  }
  if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
    unique_keys_table_range[0] = 0;
  }
}

template <typename key_t>
__global__ void hash_table_dump_index_mask_kernel(const TableEntry<key_t>* table,
                                                  const uint32_t* allgather_table_range,
                                                  int num_gpus, int num_table, int* mask) {
  for (int ith_table = blockIdx.x; ith_table < num_table; ith_table += gridDim.x) {
    uint32_t table_capacity = allgather_table_range[(ith_table + 1) * num_gpus] -
                              allgather_table_range[ith_table * num_gpus];

    auto current_table = table + allgather_table_range[ith_table * num_gpus];
    for (uint32_t i = threadIdx.x; i < table_capacity; i += blockDim.x) {
      const key_t key_hi = current_table[i].key;
      if (key_hi == 0) continue;
      const uint32_t value = current_table[i].value;
      uint32_t idx = ((value >> 1U) - 1);
      mask[idx] = 1;
    }
  }
}

__global__ void table_range_to_table_ids_and_ev_start_indices_kernel(
    int num_table, const uint32_t* table_range, const int* unique_table_ids,
    const int* table_id_to_ev_size, int* table_ids, uint32_t* ev_start_indices) {
  for (int ith_table = blockIdx.x; ith_table < num_table; ith_table += gridDim.x) {
    int table_id = unique_table_ids[ith_table];
    int ev_size = table_id_to_ev_size[table_id];

    uint32_t start = table_range[ith_table];
    uint32_t end = table_range[ith_table + 1];
    for (uint32_t i = threadIdx.x; i < (end - start); i += blockDim.x) {
      table_ids[start + i] = table_id;
      ev_start_indices[1 + start + i] = ev_size;
    }
  }
  if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
    ev_start_indices[0] = 0;
  }
}

template <typename key_t, typename HASH>
__global__ void hash_table_lookup_key_and_map_ev_start_indices(
    const key_t* local_reduce_unique_keys, const uint32_t* local_reduce_table_range, int num_table,
    int num_gpus, const TableEntry<key_t>* table, const uint32_t* allgather_table_range,
    const int* unique_idx, const uint32_t* table_range, const uint32_t* allreduce_ev_start_indices,
    uint32_t* ev_start_indices) {
  for (int ith_table = blockIdx.x; ith_table < num_table; ith_table += gridDim.x) {
    uint32_t num_keys =
        local_reduce_table_range[ith_table + 1] - local_reduce_table_range[ith_table];
    uint32_t table_capacity = allgather_table_range[(ith_table + 1) * num_gpus] -
                              allgather_table_range[ith_table * num_gpus];

    auto current_local_reduce_unique_keys =
        local_reduce_unique_keys + local_reduce_table_range[ith_table];
    auto current_table = table + allgather_table_range[ith_table * num_gpus];
    auto current_ev_start_indices = ev_start_indices + local_reduce_table_range[ith_table];
    for (uint32_t i = threadIdx.x; i < num_keys; i += blockDim.x) {
      const key_t key = current_local_reduce_unique_keys[i];
      uint32_t key_hash = HASH()(key);
      uint32_t pos = key_hash % table_capacity;
      bool prob_next = true;
      while (prob_next) {
        const key_t key_hi = current_table[pos].key;
        const uint32_t value = current_table[pos].value;
        if (key == static_cast<key_t>((key_hi & ~(0x1)) | (value & 0x1))) {
          prob_next = false;
        } else {
          pos += 1;
          if (pos >= table_capacity) {
            pos -= table_capacity;
          }
        }
      }
      const uint32_t value = current_table[pos].value;
      uint32_t idx = (value >> 1U) - 1;
      current_ev_start_indices[i] = allreduce_ev_start_indices[unique_idx[idx] - 1];
    }
  }
}

}  // namespace

void broadcast_unique_keys(const embedding::WgradEvStartIndicesCalculationInput& input,
                           BroadcastResult& broadcast_result, int num_table, int num_gpus,
                           hipStream_t stream, ncclComm_t comm) {
  // 1. collect table range
  HCTR_LIB_THROW(ncclAllGather(input.table_range.get<uint32_t>(),
                               broadcast_result.allgather_table_range_.get<uint32_t>(),
                               input.table_range.get_num_elements(), ncclUint32, comm, stream));
  input.table_range.copy_to(broadcast_result.h_table_range_, stream);

  // 2. calculate num_unique_keys on each gpu
  reorder_allgather_table_range_kernel<128><<<1, 128, 0, stream>>>(
      broadcast_result.allgather_table_range_.get<uint32_t>(), num_table, num_gpus,
      broadcast_result.reordered_allgather_table_range_.get<uint32_t>());
  broadcast_result.reordered_allgather_table_range_.copy_to(
      broadcast_result.h_reordered_allgather_table_range_, stream);
  HCTR_LIB_THROW(hipStreamSynchronize(stream));

  auto key_type = input.unique_keys.dtype();
  // 3. broadcast unique keys
  DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
    HCTR_LIB_THROW(ncclGroupStart());
    const uint32_t* h_table_range_ptr = broadcast_result.h_table_range_.get<uint32_t>();
    const uint32_t* h_reordered_allgather_table_range_ptr =
        broadcast_result.h_reordered_allgather_table_range_.get<uint32_t>();

    for (int table_id = 0; table_id < num_table; ++table_id) {
      for (int dst_gpu_id = 0; dst_gpu_id < num_gpus; ++dst_gpu_id) {
        uint32_t num_unique_keys =
            h_reordered_allgather_table_range_ptr[table_id * num_gpus + dst_gpu_id + 1] -
            h_reordered_allgather_table_range_ptr[table_id * num_gpus + dst_gpu_id];
        HCTR_LIB_THROW(ncclBroadcast(
            input.unique_keys.get<key_t>() + h_table_range_ptr[table_id],
            broadcast_result.allgather_unique_keys_.get<key_t>() +
                h_reordered_allgather_table_range_ptr[table_id * num_gpus + dst_gpu_id],
            num_unique_keys, core::get_nccl_dtype_from_tensor_scalar_type(key_type.type()),
            dst_gpu_id, comm, stream));
      }
    }
    HCTR_LIB_THROW(ncclGroupEnd());
  });
}

void unique_broadcast_result_using_hash_table(
    const embedding::WgradEvStartIndicesCalculationInput& input,
    const BroadcastResult& broadcast_result, HashTable& hash_table, Wgrad& allreduce_wgrad,
    int num_table, int num_gpus, hipStream_t stream) {
  auto key_type = input.unique_keys.dtype();
  // 4. cal table capacity range
  // 5. insert allgather_unique_keys to hash table & get allreduce_wgrad table_range
  DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
    HCTR_LIB_THROW(
        hipMemsetAsync(hash_table.hash_table_.get(), 0, hash_table.hash_table_.nbytes(), stream));
    HCTR_LIB_THROW(hipMemsetAsync(allreduce_wgrad.table_range.get(), 0,
                                   allreduce_wgrad.table_range.nbytes(), stream));
    dim3 grid_size(num_table, num_gpus, 1);
    constexpr int block_size = 512;
    auto table_ptr = reinterpret_cast<TableEntry<key_t>*>(hash_table.hash_table_.get());
    hash_table_insert_key_and_index_kernel<key_t, DirectHash><<<grid_size, block_size, 0, stream>>>(
        broadcast_result.allgather_unique_keys_.get<key_t>(),
        broadcast_result.reordered_allgather_table_range_.get<uint32_t>(), num_table, num_gpus,
        table_ptr, allreduce_wgrad.table_range.get<uint32_t>());
    size_t temp_nbytes = hash_table.d_temp_scan_table_range_storage_.nbytes();
    hipcub::DeviceScan::InclusiveSum(hash_table.d_temp_scan_table_range_storage_.get(), temp_nbytes,
                                  allreduce_wgrad.table_range.get<uint32_t>(),
                                  allreduce_wgrad.table_range.get<uint32_t>(), num_table + 1,
                                  stream);
  });
}

void cal_indices_for_sparse_allreduce(const embedding::WgradEvStartIndicesCalculationInput& input,
                                      const BroadcastResult& broadcast_result,
                                      const HashTable& hash_table,
                                      SparseAllreduceCalEVStartIndicesTempStorage& temp_storage,
                                      Wgrad& allreduce_wgrad,
                                      WgradEvStartIndicesCalculationOutput& output, int num_table,
                                      int num_gpus, hipStream_t stream) {
  auto key_type = input.unique_keys.dtype();
  //   6. select allreduce unique_keys
  DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
    HCTR_LIB_THROW(hipMemsetAsync(temp_storage.mask_unique_keys_in_allgather_unique_keys_.get(), 0,
                                   temp_storage.mask_unique_keys_in_allgather_unique_keys_.nbytes(),
                                   stream));
    auto table_ptr = reinterpret_cast<TableEntry<key_t>*>(hash_table.hash_table_.get());
    dim3 grid_size(num_table, 1, 1);
    constexpr int block_size = 512;
    hash_table_dump_index_mask_kernel<<<grid_size, block_size, 0, stream>>>(
        table_ptr, broadcast_result.reordered_allgather_table_range_.get<uint32_t>(), num_gpus,
        num_table, temp_storage.mask_unique_keys_in_allgather_unique_keys_.get<int>());
    size_t temp_nbytes = temp_storage.d_temp_select_temp_storage_.nbytes();
    hipcub::DeviceSelect::Flagged(temp_storage.d_temp_select_temp_storage_.get(), temp_nbytes,
                               broadcast_result.allgather_unique_keys_.get<key_t>(),
                               temp_storage.mask_unique_keys_in_allgather_unique_keys_.get<int>(),
                               allreduce_wgrad.unique_keys.get<key_t>(),
                               allreduce_wgrad.num_unique_keys.get<size_t>(),
                               allreduce_wgrad.unique_keys.get_num_elements(), stream);
  });

  // 7. cal allreduce wgrad table_ids & ev start indices from table_range
  DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
    // not sure if this memset can be removed
    HCTR_LIB_THROW(hipMemsetAsync(allreduce_wgrad.ev_start_indices.get(), 0,
                                   allreduce_wgrad.ev_start_indices.nbytes(), stream));
    dim3 grid_size(num_table, 1, 1);
    constexpr int block_size = 512;
    table_range_to_table_ids_and_ev_start_indices_kernel<<<grid_size, block_size, 0, stream>>>(
        num_table, allreduce_wgrad.table_range.get<uint32_t>(), input.unique_table_ids.get<int>(),
        allreduce_wgrad.attr.table_id_to_ev_size.get<int>(), allreduce_wgrad.table_ids.get<int>(),
        allreduce_wgrad.ev_start_indices.get<uint32_t>());
    size_t temp_nbytes = temp_storage.d_temp_scan_ev_start_indices_storage_.nbytes();
    hipcub::DeviceScan::InclusiveSum(temp_storage.d_temp_scan_ev_start_indices_storage_.get(),
                                  temp_nbytes, allreduce_wgrad.ev_start_indices.get<uint32_t>(),
                                  allreduce_wgrad.ev_start_indices.get<uint32_t>(),
                                  allreduce_wgrad.ev_start_indices.get_num_elements(), stream);
  });

  // 8. cal localreduce ev_start_indices
  DISPATCH_INTEGRAL_FUNCTION(key_type.type(), key_t, [&] {
    size_t temp_nbytes = temp_storage.d_temp_scan_unique_idx_temp_storage_.nbytes();
    hipcub::DeviceScan::InclusiveSum(
        temp_storage.d_temp_scan_unique_idx_temp_storage_.get(), temp_nbytes,
        temp_storage.mask_unique_keys_in_allgather_unique_keys_.get<int>(),
        temp_storage.unique_idx_.get<int>(), temp_storage.unique_idx_.get_num_elements(), stream);

    auto table_ptr = reinterpret_cast<TableEntry<key_t>*>(hash_table.hash_table_.get());
    dim3 grid_size(num_table, 1, 1);
    constexpr int block_size = 512;
    hash_table_lookup_key_and_map_ev_start_indices<key_t, DirectHash>
        <<<grid_size, block_size, 0, stream>>>(
            input.unique_keys.get<key_t>(), input.table_range.get<uint32_t>(), num_table, num_gpus,
            table_ptr, broadcast_result.reordered_allgather_table_range_.get<uint32_t>(),
            temp_storage.unique_idx_.get<int>(), allreduce_wgrad.table_range.get<uint32_t>(),
            allreduce_wgrad.ev_start_indices.get<uint32_t>(),
            output.ev_start_indices.get<uint32_t>());
  });
}

void SparseAllreduceIndexCalculation::cal_for_sparse_input(const EmbeddingInput& embedding_input,
                                                           ReductionIndices& reduction_indices,
                                                           Wgrad& local_reduce_wgrad,
                                                           Wgrad& allreduce_wgrad, int batch_size) {
  auto sparse_allreduce_cal_ev_start_indices = [&](const WgradEvStartIndicesCalculationInput& input,
                                                   WgradEvStartIndicesCalculationOutput& output,
                                                   hipStream_t stream) {
    int gpu_id = core_->get_global_gpu_id();
    int num_gpus = core_->get_global_gpu_count();
    auto comm = core_->get_nccl();

    int num_table = input.unique_table_ids.get_num_elements();

    HCTR_LIB_THROW(hipStreamSynchronize(stream));
    broadcast_unique_keys(input, cal_ev_start_indices_storage_.broadcast_result_, num_table,
                          num_gpus, stream, comm);

    unique_broadcast_result_using_hash_table(input, cal_ev_start_indices_storage_.broadcast_result_,
                                             cal_ev_start_indices_storage_.hash_table_,
                                             allreduce_wgrad, num_table, num_gpus, stream);

    cal_indices_for_sparse_allreduce(input, cal_ev_start_indices_storage_.broadcast_result_,
                                     cal_ev_start_indices_storage_.hash_table_,
                                     cal_ev_start_indices_storage_.temp_storage_, allreduce_wgrad,
                                     output, num_table, num_gpus, stream);
  };
  local_reduce_index_calculation_.cal_for_sparse_input(
      embedding_input, segmented_sort_device_, segmented_unique_, cal_dst_ids_, reduction_indices,
      local_reduce_wgrad, batch_size);
  local_reduce_index_calculation_.cal_unique_key_table_range(local_reduce_wgrad);
  local_reduce_index_calculation_.cal_dst_ev_start(local_reduce_wgrad,
                                                   sparse_allreduce_cal_ev_start_indices);
}

}  // namespace embedding
