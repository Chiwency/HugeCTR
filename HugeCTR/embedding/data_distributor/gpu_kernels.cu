#include "hip/hip_runtime.h"
#include "core/registry.hpp"
#include "gpu_kernels.hpp"
#include "include/utils.cuh"

namespace HugeCTR {

template <typename offset_t>
__global__ void compute_bucket_ranges_with_padding(offset_t* bucket_ranges,
                                                   const int* __restrict hotness_bucket_ranges,
                                                   int current_batch_size, int batch_size) {
  const int lookup = blockIdx.y;

  // e.g:
  // hotnesses:             [3, 5, 1, 2]
  // hotness_bucket_ranges: [0, 3, 8, 9, 11]
  // current_batch_size:    10
  // lookup_start_ranges:   [0,30,90,90,110]
  const int lookup_hotness_bucket_start = hotness_bucket_ranges[lookup];
  const int lookup_hotness_bucket_end = hotness_bucket_ranges[lookup + 1];
  const int lookup_hotness = lookup_hotness_bucket_end - lookup_hotness_bucket_start;
  const offset_t lookup_start_range = lookup_hotness_bucket_start * current_batch_size;
  const offset_t lookup_end_range = lookup_hotness_bucket_end * current_batch_size;

  // If we are on the last lookup, extend batch_size by 1 to account for end bucket
  int end_bucket = (lookup == gridDim.y - 1);

  CUDA_1D_KERNEL_LOOP(bucket_idx, batch_size + end_bucket) {
    const bool is_valid_bucket = bucket_idx < current_batch_size + end_bucket;
    bucket_ranges[lookup * batch_size + bucket_idx] =
        is_valid_bucket ? lookup_start_range + bucket_idx * lookup_hotness : lookup_end_range;
  }
}

void compute_fixed_bucket_ranges(core::Tensor hotness_bucket_range, int current_batch_size,
                                 int batch_size, core::Tensor bucket_range, hipStream_t stream) {
  const size_t num_lookup = hotness_bucket_range.get_num_elements() - 1;

  DISPATCH_INTEGRAL_FUNCTION(bucket_range.dtype().type(), offset_t, [&] {
    compute_bucket_ranges_with_padding<<<dim3(144 * 8, num_lookup), 256, 0, stream>>>(
        bucket_range.get<offset_t>(), hotness_bucket_range.get<int>(), current_batch_size,
        batch_size);
  });
}

}  // namespace HugeCTR